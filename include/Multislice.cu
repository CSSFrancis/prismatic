#include "hip/hip_runtime.h"
#include "Multislice.cuh"
#include "Multislice.h"
#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"




#include <iostream>
//#include "hip/hip_runtime.h"

#include "fftw3.h"
#include "utility.h"
#include "../../../../../../usr/local/cuda/include/hip/driver_types.h"

#ifdef PRISM_ENABLE_DOUBLE_PRECISION
typedef hipDoubleComplex PRISM_CUDA_COMPLEX_FLOAT;
#else
typedef hipFloatComplex PRISM_CUDA_COMPLEX_FLOAT;
#endif //PRISM_ENABLE_DOUBLE_PRECISION
#define NX 64
#define NY 64
#define NZ 128
#define PI 3.14159265359
#define BLOCK_SIZE1D 1024

// helpful function for checking CUDA errors.
// Source: http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define cudaErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define cufftErrchk(ans) { gpuAssert_cufft((ans), __FILE__, __LINE__); }
inline void gpuAssert_cufft(int code, const char *file, int line, bool abort=true){
	if (code != HIPFFT_SUCCESS)
	{
		fprintf(stderr,"GPUassert: %s %d\n", file, line);
		if (abort) exit(code);
	}
}

namespace PRISM{

	__device__ __constant__ PRISM_FLOAT_PRECISION pi       = PI;
	__device__ __constant__ PRISM_CUDA_COMPLEX_FLOAT i     = {0, 1};
	__device__ __constant__ PRISM_CUDA_COMPLEX_FLOAT pi_cx = {PI, 0};
	__device__ __constant__ PRISM_CUDA_COMPLEX_FLOAT minus_2pii = {0, -2*PI};
	__device__ __forceinline__ hipDoubleComplex exp_cx(const hipDoubleComplex a){
		double e = exp(a.x);
		double s,c;
		sincos(a.y, &s, &c);
		return make_hipDoubleComplex(e*c, e*s);
	}
	__device__ __forceinline__ hipFloatComplex exp_cx(const hipFloatComplex a){
		float e = expf(a.x);
		float s,c;
		sincosf(a.y, &s, &c);
		return make_hipFloatComplex(e*c, e*s);
	}
	__global__ void initializePsi(PRISM_CUDA_COMPLEX_FLOAT *psi_d,
	                              const PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                              const PRISM_FLOAT_PRECISION* qya_d,
	                              const PRISM_FLOAT_PRECISION* qxa_d,
	                              const size_t N,
	                              const PRISM_FLOAT_PRECISION yp,
	                              const PRISM_FLOAT_PRECISION xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		PRISM_CUDA_COMPLEX_FLOAT arg;
		arg = make_hipFloatComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmulf(PsiProbeInit_d[idx], exp_cx(hipCmulf(minus_2pii,arg)));
	}
}
	__global__ void kernel_multiply_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
	                                        const PRISM_CUDA_COMPLEX_FLOAT* other,
	                                        const size_t N){
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
			PRISM_CUDA_COMPLEX_FLOAT a = arr[idx];
			PRISM_CUDA_COMPLEX_FLOAT o = other[idx];

			arr[idx].x = a.x*o.x - a.y*o.y;
			arr[idx].y = a.x*o.y + a.y*o.x;
		}
	}
//	__global__ void kernel_multiply_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
//	                                        const PRISM_CUDA_COMPLEX_FLOAT* other,
//	                                        const size_t N){
//		int idx = threadIdx.x + blockDim.x*blockIdx.x;
//		if (idx < N) {
//			arr[idx] = hipCmulf(arr[idx], other[idx]);
//			PRISM_FLOAT_PRECISION ax = arr[idx].x;
//			PRISM_FLOAT_PRECISION ay = arr[idx].y;
//			PRISM_FLOAT_PRECISION ox = other[idx].x;
//			PRISM_FLOAT_PRECISION oy = other[idx].y;
//
//			arr[idx].x = ax*ox - ay*oy;
//			arr[idx].y = ax*oy + ay*ox;
//		}
//	}


	__global__ void kernel_divide_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
	                                        const PRISM_FLOAT_PRECISION val,
	                                        const size_t N){
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
			arr[idx].x /= val;
			arr[idx].y /= val;
		}
	}

//	__global__ void kernel_divide_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
//	                                      const PRISM_CUDA_COMPLEX_FLOAT* other,
//	                                      const size_t N){
//		int idx = threadIdx.x + blockDim.x*blockIdx.x;
//		if (idx < N) {
//			arr[idx].x *= other[idx].x;
//			arr[idx].y *= other[idx].y;
//		}
//	}

	__global__ void abs_squared(PRISM_FLOAT_PRECISION* arr,
	                            const PRISM_CUDA_COMPLEX_FLOAT* other,
	                            const size_t N){
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
//			arr[idx] = hipCabsf(other[idx]);
			PRISM_FLOAT_PRECISION re = other[idx].x;
			PRISM_FLOAT_PRECISION im = other[idx].y;
			arr[idx] = re*re + im*im;
		}
	}

	__global__ void setAll(PRISM_FLOAT_PRECISION *data, PRISM_FLOAT_PRECISION val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
	}
	__global__ void integrateDetector(const PRISM_FLOAT_PRECISION* psiabssquared_d,
	                                  const PRISM_FLOAT_PRECISION* alphaInd_d,
	                                  PRISM_FLOAT_PRECISION* integratedOutput,
	                                  const size_t N,
	                                  const size_t num_integration_bins) {
		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N) {
			size_t alpha = (size_t)alphaInd_d[idx];
			if (alpha <= num_integration_bins)
				atomicAdd(&integratedOutput[alpha-1], psiabssquared_d[idx]);
//				atomicAdd(&integratedOutput[alpha], 1);
		}
	}
  void formatOutput_gpu_integrate(Parameters<PRISM_FLOAT_PRECISION>& pars,
                                  PRISM_FLOAT_PRECISION * psiabssquared_d,
                                  const PRISM_FLOAT_PRECISION * alphaInd_d,
//                                  PRISM_FLOAT_PRECISION* integratedOutput_d,
                                  const size_t& ay,
                                  const size_t& ax,
                                  const size_t& dimj,
                                  const size_t& dimi,
                                  hipStream_t& stream){
	  PRISM_FLOAT_PRECISION * integratedOutput_d;
	  size_t num_integration_bins = pars.detectorAngles.size();
	  cudaErrchk(hipMalloc((void **) &integratedOutput_d,
	                        num_integration_bins * sizeof(PRISM_FLOAT_PRECISION)));
//	  cudaErrchk(hipMemset(integratedOutput, 1, pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
	  setAll<<< (num_integration_bins - 1)/BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(integratedOutput_d, 0, num_integration_bins);
	  integrateDetector<<< (dimj*dimi - 1)/BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(psiabssquared_d, alphaInd_d, integratedOutput_d, dimj*dimi, num_integration_bins);

	  cudaErrchk(hipMemcpy(&pars.stack.at(ay,ax,0,0),integratedOutput_d,
	                        num_integration_bins * sizeof(PRISM_FLOAT_PRECISION),
	                        hipMemcpyDeviceToHost));

	  cudaErrchk(hipFree(integratedOutput_d));
//	  int a;

}

__host__ void getMultisliceProbe_gpu(Parameters<PRISM_FLOAT_PRECISION>& pars,
									 PRISM_CUDA_COMPLEX_FLOAT* trans_d,
									 PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
//									 PRISM_FLOAT_PRECISION* integratedOutput_d,
									 const PRISM_FLOAT_PRECISION* qya_d,
									 const PRISM_FLOAT_PRECISION* qxa_d,
									 const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
									 const size_t& ay,
									 const size_t& ax,
									 const size_t dimj,
									 const size_t dimi,
									 const PRISM_FLOAT_PRECISION* alphaInd_d,
									 hipStream_t& stream){
//		cout << "stream = " << stream << endl;
//		cout << "ax = " << ax <<endl;
//		cout << "ay = " << ay <<endl;
		// create cuFFT plan
		hipfftHandle plan;
		cufftErrchk(hipfftPlan2d(&plan, dimi, dimj, HIPFFT_C2C));

//		hipStream_t stream2;
//		cudaErrchk(hipStreamCreate(&stream2));
		// set the stream
//		cufftErrchk(hipfftSetStream(plan, stream2));
		cufftErrchk(hipfftSetStream(plan, stream));

		// initialize psi
		PRISM_FLOAT_PRECISION yp = pars.yp[ay];
		PRISM_FLOAT_PRECISION xp = pars.xp[ax];
		PRISM_CUDA_COMPLEX_FLOAT *psi_d;
		PRISM_FLOAT_PRECISION *psi_abssquared_d;
		const size_t N = dimj*dimi;
		cudaErrchk(hipMalloc((void**)&psi_d, dimj*dimi*sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
		cudaErrchk(hipMalloc((void**)&psi_abssquared_d, dimj*dimi*sizeof(PRISM_FLOAT_PRECISION)));
//		cout << " xp = " << xp << endl;
//		cout << " yp = " << yp << endl;


		//initializePsi<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);
		initializePsi<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);

//		initializePsi<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D>>>(psi_d, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);
//
//
//		for (auto planeNum = 0; planeNum < 1; ++planeNum) {
//			hipfftExecC2C(plan, psi_d, psi_d, HIPFFT_BACKWARD);
//			kernel_multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream2>>>(psi_d, &trans_d[planeNum*N], N);
//			hipfftExecC2C(plan, psi_d, psi_d, HIPFFT_FORWARD);
//			kernel_multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream2>>>(psi_d, prop_d, N);
//			kernel_divide_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream2>>>(psi_d, N, N);
//		}


		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(hipfftExecC2C(plan, &psi_d[0], &psi_d[0], HIPFFT_BACKWARD));
			kernel_multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, &trans_d[planeNum*N], N);
			cufftErrchk(hipfftExecC2C(plan, &psi_d[0], &psi_d[0], HIPFFT_FORWARD));
			kernel_multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, prop_d, N);
			kernel_divide_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, N, N);
		}

		abs_squared<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_abssquared_d, psi_d, N);
		//hipMemcpyAsync(output, psi_abssquared_d,N*sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost,stream2);
//	cudaErrchk(hipMemcpy(output, psi_abssquared_d,N*sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost));


	formatOutput_gpu_integrate(pars, psi_abssquared_d, alphaInd_d, ay, ax, dimj, dimi,stream);

//		if(ay==0 && ax==0) {
//			for (auto j = 0; j < 25; ++j)cout << "output[j] = " << output[j] << endl;
////		Array2D<PRISM_FLOAT_PRECISION> = zeros_ND<2, PRISM_FLOAT_PRECISION>({{dimj, dimi}});
//		}
//
//	std::complex<PRISM_FLOAT_PRECISION> answer;
//
////	hipMemcpy(&answer, psi_d+(ax),1*sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost);
//////	cout << " answer = " << answer << endl;
//		if (ax==0 && ay==0) {
//			for (auto j = 0; j < 10; ++j) {
//				cudaErrchk(hipMemcpy(&answer, psi_d + j, 1 * sizeof(PRISM_CUDA_COMPLEX_FLOAT), hipMemcpyDeviceToHost));
//				cout << " psi_d = " << answer << endl; //<< "xp = " << xp << "yp = " << yp << endl;
//			}
//		}
//
//		PRISM_FLOAT_PRECISION answer2;
//
////	hipMemcpy(&answer, psi_d+(ax),1*sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost);
//////	cout << " answer = " << answer << endl;
//		if (ax==0 && ay==0) {
//			for (auto j = 0; j < 10; ++j) {
//				cudaErrchk(hipMemcpy(&answer2, psi_abssquared_d + j, 1 * sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost));
//				cout << " psi_abssquared_d = " << answer2 << endl << "xp = " << xp << "yp = " << yp << endl;
//			}
//		}
//
//		std::complex<PRISM_FLOAT_PRECISION> answer_propd;
//
////	hipMemcpy(&answer, psi_d+(ax),1*sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost);
//////	cout << " answer = " << answer << endl;
//		if (ax==0 && ay==0) {
//			for (auto j = 0; j < 10; ++j) {
//				cudaErrchk(hipMemcpy(&answer_propd, prop_d + j, 1 * sizeof(std::complex<PRISM_FLOAT_PRECISION>), hipMemcpyDeviceToHost));
//				cout << " prop_d = " << answer_propd << endl;
//			}
//		}
//
//		std::complex<PRISM_FLOAT_PRECISION> answer_trans_d;
//
////	hipMemcpy(&answer, psi_d+(ax),1*sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost);
//////	cout << " answer = " << answer << endl;
//		if (ax==0 && ay==0) {
//			for (auto j = 0; j < 10; ++j) {
//				cudaErrchk(hipMemcpy(&answer_trans_d, trans_d + j, 1 * sizeof(std::complex<PRISM_FLOAT_PRECISION>), hipMemcpyDeviceToHost));
//				cout << " answer_trans_d = " << answer_trans_d << endl;
//			}
//		}
//
//

/*
psi(:) = PsiProbeInit ...
		.* exp(-2i*pi ...
		*(emdSTEM.qxa*emdSTEM.MULTIxp(a0) ...
		+ emdSTEM.qya*emdSTEM.MULTIyp(a1)));


copy PsiProbeInit,qxa, qya to GPU hipComplex types. Allocate a pinned output that is same type as stack
run kernel to accomplish above line. If necessary, write a hipComplex mult(const hipComplex& a,const hipComplex& b) function and a hipComplex my_exp(const hipComplex& a,const hipComplex& b) that uses exp(a + bi) = exp(a) *cos(a) + exp(a) * sin(a)*i

create two cufft plans, write a complex inplace multiplication kernel, and apply the following loop
for a2 = 1:emdSTEM.numPlanes
		psi = fft2(ifft2(psi).*trans(:,:,a2)).*emdSTEM.prop;
	end

take final FFT and abs square

async stream the result to the pinned stack

after done copy the pinned stack to original

*/
	cufftErrchk(hipfftDestroy(plan));
	cudaErrchk(hipFree(psi_d));
	cudaErrchk(hipFree(psi_abssquared_d));
//	cudaErrchk(hipStreamDestroy(stream2));
}
    __host__ void buildMultisliceOutput_gpu(Parameters <PRISM_FLOAT_PRECISION> &pars,
                                            Array3D <std::complex<PRISM_FLOAT_PRECISION>> &trans,
                                            Array2D <std::complex<PRISM_FLOAT_PRECISION>> &PsiProbeInit,
                                            Array2D <PRISM_FLOAT_PRECISION> &alphaInd) {
		cout << "debug pars.prop" << endl;
		for (auto i =0; i < 10; ++i)cout << pars.prop[i] << endl;
		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		using namespace std;
		cout << "DEBUG PsiProbeInit.at(0,1) = " << PsiProbeInit.at(0,1) << endl;
		cout << "DEBUG PsiProbeInit[1] = " << PsiProbeInit[1] << endl;
		cout << "DEBUG PsiProbeInit[2] = " << PsiProbeInit[2] << endl;

		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		hipStream_t streams[total_num_streams];
		cout <<"total_num_streams = " << total_num_streams<< endl;
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&streams[j]));
		}
		cout << "first stream = " << streams[0] << endl;
        cout << "Test GPU function from CUDA host" << endl;
		const PRISM_FLOAT_PRECISION cpu_stop = std::floor(pars.meta.cpu_gpu_ratio*pars.yp.size());
		vector<thread> workers_gpu;
		vector<thread> workers_cpu;
		workers_gpu.reserve(total_num_streams); // prevents multiple reallocations
		workers_cpu.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations

		// pointers to GPU memory (one copy per GPU)
		PRISM_CUDA_COMPLEX_FLOAT *PsiProbeInit_d[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT *trans_d[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT *prop_d[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    *qxa_d[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    *qya_d[pars.meta.NUM_GPUS];
	    PRISM_FLOAT_PRECISION    *alphaInd_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *integratedOutput_d[pars.meta.NUM_GPUS];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &PsiProbeInit_d[g],     PsiProbeInit.size()        * sizeof(PsiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &trans_d[g],            trans.size()               * sizeof(trans[0])));
			cudaErrchk(hipMalloc((void **) &qxa_d[g],              pars.qxa.size()            * sizeof(pars.qxa[0])));
			cudaErrchk(hipMalloc((void **) &qya_d[g],              pars.qya.size()            * sizeof(pars.qya[0])));
			cudaErrchk(hipMalloc((void **) &prop_d[g],             pars.prop.size()           * sizeof(pars.prop[0])));
			cudaErrchk(hipMalloc((void **) &alphaInd_d[g],         alphaInd.size()            * sizeof(alphaInd[0])));
//			cudaErrchk(hipMalloc((void **) &integratedOutput_d[g], pars.detectorAngles.size() * sizeof(alphaInd[0])));
		}

		// copy memory to each GPU (this can be made asynchronous if necessary by copying to pinned memory first)
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpy(PsiProbeInit_d[g], &PsiProbeInit[0],
			                      PsiProbeInit.size() * sizeof(PsiProbeInit[0]), hipMemcpyHostToDevice));
			cudaErrchk(hipMemcpy(trans_d[g], &trans[0],
			                      trans.size() * sizeof(trans[0]), hipMemcpyHostToDevice));
			cudaErrchk(hipMemcpy(qxa_d[g], &pars.qxa[0],
			                      pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice));
			cudaErrchk(hipMemcpy(qya_d[g], &pars.qya[0],
			                      pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice));
			cudaErrchk(hipMemcpy(prop_d[g], &pars.prop[0],
			                      pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice));
			cudaErrchk(hipMemcpy(alphaInd_d[g], &alphaInd[0],
			                      alphaInd.size() * sizeof(alphaInd[0]), hipMemcpyHostToDevice));
		}
//
//		std::complex<float> b;
//		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
//			hipSetDevice(g);
//			hipMemcpy(&b, PsiProbeInit_d[g]+1, 1 * sizeof(PsiProbeInit[0]), hipMemcpyDeviceToHost);
//			cout << "this b = " << b << endl;
//		}

		size_t psi_size = PsiProbeInit.size();

		auto WORK_CHUNK_SIZE_GPU = std::floor( (((pars.yp.size() - cpu_stop - 1)) / total_num_streams) + 1); //TODO: divide work more generally than just splitting up by yp. If input isn't square this might not do a good job
		cout << "WORK_CHUNK_SIZE_GPU = " << WORK_CHUNK_SIZE_GPU << endl;
		auto start = cpu_stop;// gpu work starts where the cpu work will stop
		auto stop = start + WORK_CHUNK_SIZE_GPU;
		int stream_count = 0;
		while (start < pars.yp.size()) {
			int gpu_num = stream_count % pars.meta.NUM_GPUS; // determine which gpu handles this job
			hipStream_t& current_stream = streams[stream_count];
			cout << " STREAM = " << streams[stream_count] << endl;
			cout << "Launching thread to compute all x-probe positions for y-probes "
				 << start << "-" << std::min((size_t)stop,pars.yp.size()) << " on stream #" << stream_count << " of GPU #" << gpu_num << '\n';

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = PsiProbeInit_d[gpu_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_d = trans_d[gpu_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d  = prop_d[gpu_num];
			PRISM_FLOAT_PRECISION *current_qxa_d      = qxa_d[gpu_num];
			PRISM_FLOAT_PRECISION *current_qya_d      = qya_d[gpu_num];
			PRISM_FLOAT_PRECISION *current_alphaInd_d = alphaInd_d[gpu_num];
//			PRISM_FLOAT_PRECISION *current_integratedOutput_d = integratedOutput_d[gpu_num];

			// launch a new thread
			// emplace_back is better whenever constructing a new object
			workers_gpu.emplace_back(thread([&pars, current_trans_d, current_PsiProbeInit_d, &alphaInd, current_alphaInd_d,
					                                start, stop, gpu_num, current_qya_d, current_qxa_d,
					                                current_prop_d, &current_stream, &psi_size, &PsiProbeInit]() {

				// page-locked (pinned) memory for async streaming of the result back
//				PRISM_FLOAT_PRECISION *pinned_output;

				// figure out how much pinned memory to allocate in this job
				size_t num_probe_positions = pars.xp.size() * (min((size_t) stop, pars.yp.size()) - start);
//				size_t pinned_output_size = psi_size * num_probe_positions;

				// allocate pinned memory on host
//				cout << "pinned_output_size * sizeof(PRISM_FLOAT_PRECISION)) = " << pinned_output_size * sizeof(PRISM_FLOAT_PRECISION) << endl;
//				cout << "num_probe_positions = " << num_probe_positions<< endl;
//				cout << "psi_size = " << psi_size << endl;
//				cout << "PsiProbeInit.get_dimj() = " << PsiProbeInit.get_dimj() <<endl;
//				cout << "PsiProbeInit.get_dimi() = " << PsiProbeInit.get_dimi() <<endl;
//				cudaErrchk(hipHostMalloc((void **) &pinned_output, pinned_output_size * sizeof(PRISM_FLOAT_PRECISION)));

				// set the GPU context
				cudaErrchk(hipSetDevice(gpu_num)); // set current gpu
//				PRISM_FLOAT_PRECISION *pinned_output_begin = pinned_output; // pointer to the beginning of corresponding output layer in the 3D array
				for (auto ay = start; ay < std::min((size_t) stop, pars.yp.size()); ++ay) {
					for (auto ax = 0; ax < pars.xp.size(); ++ax) {
//				for (auto ay = start; ay < 25; ++ay) {
//					for (auto ax = 0; ax < 25; ++ax) {
						getMultisliceProbe_gpu(pars, current_trans_d, current_PsiProbeInit_d, current_qya_d,
						                       current_qxa_d,
						                       current_prop_d, ay, ax, PsiProbeInit.get_dimj(), PsiProbeInit.get_dimi(),
						                       current_alphaInd_d, current_stream);
//						pinned_output_begin += psi_size; // advance the start point of the output
					}
				}

//				cudaErrchk(hipDeviceSynchronize());
//				{
//					for (auto j = 0; j < 25; ++j)cout << "pinned_output[j] = " << pinned_output[j] << endl;
//				}
//				PRISM_FLOAT_PRECISION c = 0;
//				for (auto i = 0; i < pinned_output_size; ++i) {
//					c += pinned_output[i];
//				}
//				cout << "c = " << c << endl;
//				PRISM_FLOAT_PRECISION *counts = pinned_output;

//				Array2D<PRISM_FLOAT_PRECISION> db = zeros_ND<2, PRISM_FLOAT_PRECISION>({{PsiProbeInit.get_dimj(), PsiProbeInit.get_dimi()}});
//				auto db_ptr = db.begin();
//				for (auto ay = start; ay < 25; ++ay) {
//					for (auto ax = 0; ax < 25; ++ax) {
//				for (auto ay = start; ay < std::min((size_t) stop, pars.yp.size()); ++ay) {
//					for (auto ax = 0; ax < pars.xp.size(); ++ax) {
//						auto idx = alphaInd.begin();
//						while (idx != alphaInd.end()) {
//							if (ay==0 && ax==0)*db_ptr = *counts;
//							if (*idx <= pars.Ndet) {
//								cout << "count = " << *counts << endl;
//								pars.stack.at(ay, ax, (*idx) - 1, 0) += (*counts);

//								cout << "ax = " << ax << endl;
//								cout << "ay = " << ay << endl;
//								cout << "(*idx) - 1 = " << (*idx) - 1 << endl;
//								cout << "pars.stack.at(ay, ax, (*idx) - 1, 0) = "
//								     << pars.stack.at(ay, ax, (*idx) - 1, 0)
//								     << endl;
//							}
//							db_ptr++;
//							++idx;
//							++counts;
//						}
//					}
//				}
//				db.toMRC_f("db_intOutput.mrc");
//			cout << " DEBUG 1 " << endl;
//				if (start == 0 ) {
//					Array2D <PRISM_FLOAT_PRECISION> prism_image;
//					prism_image = zeros_ND<2, PRISM_FLOAT_PRECISION>({{pars.stack.get_diml(), pars.stack.get_dimk()}});
//					for (auto y = 0; y < pars.stack.get_diml(); ++y) {
//						for (auto x = 0; x < pars.stack.get_dimk(); ++x) {
//							for (auto b = 13; b < 18; ++b) {
//								prism_image.at(y, x) += pars.stack.at(y, x, b, 0);
////								cout << "prism_image.at(y, x) = " << prism_image.at(y, x) << endl;
//							}
//						}
//					}
////					prism_image.toMRC_f("TEST.mrc");
//					cout <<" debug written" <<endl;
//				}
//				for (auto y = 0; y < PsiProbeInit.get_dimj(),; ++y) {
//					for (auto x = 0; x < PsiProbeInit.get_dimi(),; ++x) {
//						auto idx = alphaInd.at(y,x);
//						if ( alphaInd.at(y,x) <= pars.Ndet){
//							pars.stack.at(ay,ax,(*idx)-1, 0) += *counts * pars.scale;
//						}
//					}
//				}
//				auto idx = alphaInd.begin();
//				for (auto counts = intOutput.begin(); counts != intOutput.end(); ++counts){
//					if (*idx <= pars.Ndet){
//						pars.stack.at(ay,ax,(*idx)-1, 0) += *counts * pars.scale;
//					}
//					++idx;
//				};

//				auto stack_ptr = &pars.stack[start*];
//				cudaErrchk(hipHostFree(pinned_output));
			}));

			stream_count++;
			start += WORK_CHUNK_SIZE_GPU;
			if (start >= pars.yp.size())break;
			stop += WORK_CHUNK_SIZE_GPU;
		}


		// now launch CPU work
		auto WORK_CHUNK_SIZE_CPU = std::floor(((cpu_stop - 1) / pars.meta.NUM_THREADS) + 1); //TODO: divide work more generally than just splitting up by yp. If input isn't square this might not do a good job
		cout << "WORK_CHUNK_SIZE_CPU = " << WORK_CHUNK_SIZE_CPU << endl;
                start = 0;// cpu work starts at beginning
                stop = start + WORK_CHUNK_SIZE_CPU;
                while (start < cpu_stop) {
                        cout << "Launching thread to compute all x-probe positions for y-probes "
                                 << start << "-" << std::min(stop,cpu_stop) << " on CPU\n";
                        // emplace_back is better whenever constructing a new object
                        workers_cpu.emplace_back(thread([&pars, &trans,
                                                                                                &alphaInd, &PsiProbeInit,
                                                                                                start, cpu_stop,stop]() {
                                for (auto ay = start; ay < std::min(stop, cpu_stop); ++ay) {
                                        for (auto ax = 0; ax < pars.xp.size(); ++ax) {
                                                getMultisliceProbe_cpu(pars, trans, PsiProbeInit, ay, ax, alphaInd);
                                        }
                                }
                        }));
                        start += WORK_CHUNK_SIZE_CPU;
                        if (start >= cpu_stop)break;
                        stop += WORK_CHUNK_SIZE_CPU;
                }

		// synchronize threads
		cout << "waiting on threads" << endl;
		for (auto& t:workers_gpu)t.join();
		for (auto& t:workers_cpu)t.join();
		cout << "threads done, cleaning up" << endl;
		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j){
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceSynchronize());
			cudaErrchk(hipFree(PsiProbeInit_d[j]));
			cudaErrchk(hipFree(trans_d[j]));
			cudaErrchk(hipFree(qxa_d[j]));
			cudaErrchk(hipFree(qya_d[j]));
			cudaErrchk(hipFree(prop_d[j]));
			cudaErrchk(hipFree(alphaInd_d[j]));
//			cudaErrchk(hipFree(integratedOutput_d[j]));
		}
		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j)cudaErrchk(hipStreamDestroy(streams[j]));
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}
	}
}
