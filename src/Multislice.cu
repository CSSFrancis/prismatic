#include "hip/hip_runtime.h"
// Calculate result of Multislice simulation using GPU and (potentially) CPU. Multiple GPU threads are launched, each with
// their own memory buffers. Page-locked host memory is allocated so that memory transfers to the GPU can occur asynchronously,
// and memory allocation for the GPU occurs only once, as each call to hipMalloc will potentially interrupt concurrent execution.
// Each GPU/CPU worker thread repeatedly calls getWorkID to be assigned probe positions to compute. This queue mechanism
// ensures that both the CPU and GPU are kept busy.

// For variable naming, the suffixes are "_d" for "device" (1 copy per GPU), "_ds" for "device stream (1 copy per stream), "_ph" for "pinned host"

#include "Multislice.cuh"
#include "Multislice.h"
#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"
#include "getWorkID.h"
#include <iostream>
#include "fftw3.h"
#include "utility.h"


#define PI 3.14159265359
#define BLOCK_SIZE1D 1024

//// helpful function for checking CUDA errors.
//// Source: http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
//#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
//inline void GPUAssert(hipError_t code, const char *file, int line, bool abort=true){
//	if (code != hipSuccess)
//	{
//		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//		if (abort) exit(code);
//	}
//}
//
//// helpful function for checking cuFFT errors
//#define cufftErrchk(ans) { GPUAssert_cufft((ans), __FILE__, __LINE__); }
//inline void GPUAssert_cufft(int code, const char *file, int line, bool abort=true){
//	if (code != HIPFFT_SUCCESS)
//	{
//		fprintf(stderr,"GPUassert: %s %d\n", file, line);
//		if (abort) exit(code);
//	}
//}

namespace PRISM{
	// define some constants
	__device__ __constant__ PRISM_FLOAT_PRECISION pi       = PI;
	__device__ __constant__ PRISM_CUDA_COMPLEX_FLOAT i     = {0, 1};
	__device__ __constant__ PRISM_CUDA_COMPLEX_FLOAT pi_cx = {PI, 0};
	__device__ __constant__ PRISM_CUDA_COMPLEX_FLOAT minus_2pii = {0, -2*PI};
	
	// computes exp(real(a) + i * imag(a))
	__device__ __forceinline__ hipDoubleComplex exp_cx(const hipDoubleComplex a){
		double e = exp(a.x);
		double s,c;
		sincos(a.y, &s, &c);
		return make_hipDoubleComplex(e*c, e*s);
	}
	__device__ __forceinline__ hipFloatComplex exp_cx(const hipFloatComplex a){
		float e = expf(a.x);
		float s,c;
		sincosf(a.y, &s, &c);
		return make_hipFloatComplex(e*c, e*s);
	}
	
	// creates initial probe using existing GPU memory rather than streaming each probe
	__global__ void initializePsi(PRISM_CUDA_COMPLEX_FLOAT *psi_d,
	                              const PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                              const PRISM_FLOAT_PRECISION* qya_d,
	                              const PRISM_FLOAT_PRECISION* qxa_d,
	                              const size_t N,
	                              const PRISM_FLOAT_PRECISION yp,
	                              const PRISM_FLOAT_PRECISION xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
			PRISM_CUDA_COMPLEX_FLOAT arg;
			arg = (PRISM_CUDA_COMPLEX_FLOAT)make_hipFloatComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
			psi_d[idx] = hipCmulf(PsiProbeInit_d[idx], exp_cx(hipCmulf(minus_2pii,arg)));
		}
	}
	
	// multiply two complex arrays
	__global__ void multiply_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
	                                 const PRISM_CUDA_COMPLEX_FLOAT* other,
	                                 const size_t N){
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
			PRISM_CUDA_COMPLEX_FLOAT a = arr[idx];
			PRISM_CUDA_COMPLEX_FLOAT o = other[idx];
			arr[idx].x = a.x * o.x - a.y * o.y;
			arr[idx].y = a.x * o.y + a.y * o.x;
		}
	}

	// divide two complex arrays
	__global__ void divide_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
	                                        const PRISM_FLOAT_PRECISION val,
	                                        const size_t N){
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
			arr[idx].x /= val;
			arr[idx].y /= val;
		}
	}

	// compute modulus squared of other and store in arr
	__global__ void abs_squared(PRISM_FLOAT_PRECISION* arr,
	                            const PRISM_CUDA_COMPLEX_FLOAT* other,
	                            const size_t N){
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx < N) {
			PRISM_FLOAT_PRECISION re = other[idx].x;
			PRISM_FLOAT_PRECISION im = other[idx].y;
			arr[idx] = re*re + im*im;
		}
	}

	// set all array values to val
	__global__ void setAll(PRISM_FLOAT_PRECISION *data, PRISM_FLOAT_PRECISION val, size_t N) {
		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx<N) {
			data[idx] = val;
		}
	}

	// integrate computed intensities radially
	__global__ void integrateDetector(const PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                  const PRISM_FLOAT_PRECISION* alphaInd_d,
	                                  PRISM_FLOAT_PRECISION* integratedOutput,
	                                  const size_t N,
	                                  const size_t num_integration_bins) {
		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N) {
			size_t alpha = (size_t)alphaInd_d[idx];
			if (alpha <= num_integration_bins)
				atomicAdd(&integratedOutput[alpha-1], psi_intensity_ds[idx]);
		}
	}

	// formatOutput variants control how the resulting calculation is returned.
	// formatOutput_GPU_integrate integrates the result of the calculation at the detector plane radially and
	// asynchronously streams it back to the host pinned memory buffer, where it is copied to the final stack
    void formatOutput_GPU_integrate(Parameters<PRISM_FLOAT_PRECISION> &pars,
                                    PRISM_FLOAT_PRECISION *psi_intensity_ds,
                                    const PRISM_FLOAT_PRECISION *alphaInd_d,
                                    PRISM_FLOAT_PRECISION *stack_ph,
                                    PRISM_FLOAT_PRECISION *integratedOutput_ds,
                                    const size_t& ay,
                                    const size_t& ax,
                                    const size_t& dimj,
                                    const size_t& dimi,
                                    hipStream_t& stream){
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	  size_t num_integration_bins = pars.detectorAngles.size();
	  setAll<<< (num_integration_bins - 1)/BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(integratedOutput_ds, 0, num_integration_bins);
	  integrateDetector<<< (dimj*dimi - 1)/BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(psi_intensity_ds, alphaInd_d, integratedOutput_ds, dimj*dimi, num_integration_bins);

	  // Copy result. For the integration case the 4th dim of stack is 1, so the offset strides need only consider k and j
	  cudaErrchk(hipMemcpyAsync(&stack_ph[ay*pars.stack.get_dimk()*pars.stack.get_dimj()+ ax*pars.stack.get_dimj()],integratedOutput_ds,
	                        num_integration_bins * sizeof(PRISM_FLOAT_PRECISION),
	                        hipMemcpyDeviceToHost, stream));

	  // wait for the copy to complete and then copy on the host. Other host threads exist doing work so this wait isn't costing anything
	  cudaErrchk(hipStreamSynchronize(stream));
//	  hipDeviceSynchronize();
//		volatile long a = 0;
//		for (volatile long long b = 0; b < 1000000; ++b)
//		{
//			a += b;
//		}
	  const size_t stack_start_offset = ay*pars.stack.get_dimk()*pars.stack.get_dimj()+ ax*pars.stack.get_dimj();
	  memcpy(&pars.stack[stack_start_offset], &stack_ph[stack_start_offset], num_integration_bins * sizeof(PRISM_FLOAT_PRECISION));
}

	// computes the result of probe position ay,ax using the GPU. The effect of this function is the same as getMultisliceProbe_CPU
	__host__ void getMultisliceProbe_GPU(Parameters<PRISM_FLOAT_PRECISION>& pars,
									    PRISM_CUDA_COMPLEX_FLOAT* trans_d,
									    PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
									    PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
									    PRISM_FLOAT_PRECISION* stack_ph,
									    PRISM_FLOAT_PRECISION* psi_intensity_ds,
									    PRISM_FLOAT_PRECISION* integratedOutput_ds,
									    const PRISM_FLOAT_PRECISION* qya_d,
									    const PRISM_FLOAT_PRECISION* qxa_d,
									    const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
									    const size_t& ay,
									    const size_t& ax,
									    const size_t dimj,
									    const size_t dimi,
									    const PRISM_FLOAT_PRECISION* alphaInd_d,
									    const hipfftHandle& plan,
									    hipStream_t& stream){

		// initialize psi
		PRISM_FLOAT_PRECISION yp = pars.yp[ay];
		PRISM_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t N = dimj*dimi;
		initializePsi<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);

		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(hipfftExecC2C(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, &trans_d[planeNum*N], N);
			cufftErrchk(hipfftExecC2C(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, N);
			divide_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, N, N);
		}
		abs_squared<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_intensity_ds, psi_ds, N);
		formatOutput_GPU_integrate(pars, psi_intensity_ds, alphaInd_d, stack_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
}
    __host__ void buildMultisliceOutput_GPU(Parameters <PRISM_FLOAT_PRECISION> &pars,
                                            Array3D <std::complex<PRISM_FLOAT_PRECISION>> &trans,
                                            Array2D <std::complex<PRISM_FLOAT_PRECISION>> &PsiProbeInit,
                                            Array2D <PRISM_FLOAT_PRECISION> &alphaInd) {


		cudaErrchk(hipDeviceReset());
		cudaErrchk(hipFree(0));
		cout << "debug pars.prop" << endl;
		cout << "Psi dim y = " << PsiProbeInit.get_dimj() << endl;
		cout << "Psi dim x = " << PsiProbeInit.get_dimi() << endl;
		cout << "number of planes = " << pars.numPlanes << endl;
		for (auto i =0; i < 10; ++i)cout << pars.prop[i] << endl;
		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		using namespace std;

		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		hipStream_t streams[total_num_streams];
		hipfftHandle cufft_plan[total_num_streams];
		cout <<"total_num_streams = " << total_num_streams<< endl;
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&streams[j]));
			cufftErrchk(hipfftPlan2d(&cufft_plan[j], PsiProbeInit.get_dimi(), PsiProbeInit.get_dimj(), HIPFFT_C2C));
			cufftErrchk(hipfftSetStream(cufft_plan[j], streams[j]));
		}


		vector<thread> workers_GPU;
		vector<thread> workers_CPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations
		workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations

		// pointers to pinned host memory for async transfers
		std::complex<PRISM_FLOAT_PRECISION>  *PsiProbeInit_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *trans_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *prop_ph;
		PRISM_FLOAT_PRECISION                *qxa_ph;
		PRISM_FLOAT_PRECISION                *qya_ph;
		PRISM_FLOAT_PRECISION                *alphaInd_ph;
		PRISM_FLOAT_PRECISION                *stack_ph;

		// allocate pinned memory
		cudaErrchk(hipHostMalloc((void **)&PsiProbeInit_ph, PsiProbeInit.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&trans_ph,        trans.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&prop_ph,         pars.prop.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&qxa_ph,          pars.qxa.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&qya_ph,          pars.qya.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&alphaInd_ph,     alphaInd.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&stack_ph,        pars.stack.size()*sizeof(PRISM_FLOAT_PRECISION)));

		// copy host memory to pinned
		memcpy(PsiProbeInit_ph, &PsiProbeInit[0], PsiProbeInit.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(trans_ph,        &trans[0],        trans.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(prop_ph,         &pars.prop[0],    pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(qxa_ph,          &pars.qxa[0],     pars.qxa.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(qya_ph,          &pars.qya[0],     pars.qya.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(alphaInd_ph,     &alphaInd[0],     alphaInd.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(stack_ph,        &pars.stack[0],   pars.stack.size() * sizeof(PRISM_FLOAT_PRECISION));




		// pointers to read-only GPU memory (one copy per GPU)
		PRISM_CUDA_COMPLEX_FLOAT *PsiProbeInit_d[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT *trans_d[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT *prop_d[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    *qxa_d[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    *qya_d[pars.meta.NUM_GPUS];
	    PRISM_FLOAT_PRECISION    *alphaInd_d[pars.meta.NUM_GPUS];

		// pointers to read/write GPU memory (one per stream)
		PRISM_CUDA_COMPLEX_FLOAT *psi_ds[total_num_streams];
		PRISM_FLOAT_PRECISION    *psi_intensity_ds[total_num_streams];
		PRISM_FLOAT_PRECISION    *integratedOutput_ds[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &PsiProbeInit_d[g],     PsiProbeInit.size()        * sizeof(PsiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &trans_d[g],            trans.size()               * sizeof(trans[0])));
			cudaErrchk(hipMalloc((void **) &prop_d[g],             pars.prop.size()           * sizeof(pars.prop[0])));
			cudaErrchk(hipMalloc((void **) &qxa_d[g],              pars.qxa.size()            * sizeof(pars.qxa[0])));
			cudaErrchk(hipMalloc((void **) &qya_d[g],              pars.qya.size()            * sizeof(pars.qya[0])));
			cudaErrchk(hipMalloc((void **) &alphaInd_d[g],         alphaInd.size()            * sizeof(alphaInd[0])));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &psi_ds[s],              PsiProbeInit.size()        * sizeof(PsiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &psi_intensity_ds[s],    PsiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &integratedOutput_ds[s], pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(psi_ds[s], 0, PsiProbeInit.size()        * sizeof(PsiProbeInit[0])));
			cudaErrchk(hipMemset(psi_intensity_ds[s], 0, PsiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(integratedOutput_ds[s], 0, pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
		}


		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(PsiProbeInit_d[g], &PsiProbeInit_ph[0],
			                      PsiProbeInit.size() * sizeof(PsiProbeInit[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(trans_d[g], &trans_ph[0],
			                      trans.size() * sizeof(trans[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(prop_d[g], &prop_ph[0],
			                      pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(qxa_d[g], &qxa_ph[0],
			                      pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(qya_d[g], &qya_ph[0],
			                      pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(alphaInd_d[g], &alphaInd_ph[0],
			                      alphaInd.size() * sizeof(alphaInd[0]), hipMemcpyHostToDevice, streams[stream_id]));
		}


		size_t psi_size = PsiProbeInit.size();
		int stream_count = 0;
		setWorkStartStop(0, pars.xp.size() * pars.yp.size());
		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipStream_t& current_stream = streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << '\n';

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = PsiProbeInit_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_d = trans_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d  = prop_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qxa_d      = qxa_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qya_d      = qya_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_alphaInd_d = alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds           = psi_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_psi_intensity_ds    = psi_intensity_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_integratedOutput_ds = integratedOutput_ds[stream_count];
			hipfftHandle & current_cufft_plan = cufft_plan[stream_count];
			// launch a new thread
			// emplace_back is better whenever constructing a new object
			workers_GPU.emplace_back(thread([&pars, current_trans_d, current_PsiProbeInit_d, &alphaInd, current_alphaInd_d,
					                                current_psi_ds, current_psi_intensity_ds, current_integratedOutput_ds,
					                                GPU_num, current_qya_d, current_qxa_d,stack_ph,current_cufft_plan,
					                                current_prop_d, &current_stream, &psi_size, &PsiProbeInit, stream_count]() {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU
				size_t Nstart, Nstop, ay, ax;
				while (getWorkID(pars, Nstart, Nstop)){ // synchronously get work assignment
					while (Nstart != Nstop){
						ay = Nstart / pars.xp.size();
						ax = Nstart % pars.xp.size();
						if (ax==0 & ay==0) {
							cout << "ON GPU" << endl;
							cout << "pars.stack.at(0,0,0) =  " << pars.stack.at(0, 0, 0) << endl;

							complex<float> deb2;
							for (auto jj = 0; jj < 100; ++jj) {
								hipMemcpy(&deb2, current_psi_ds + jj, sizeof(PRISM_FLOAT_PRECISION),
								           hipMemcpyDeviceToHost);
								cout << "psi_ds before = " << deb2 << endl;
							}
						}
							getMultisliceProbe_GPU(pars, current_trans_d, current_PsiProbeInit_d, current_psi_ds, stack_ph,
						                       current_psi_intensity_ds,
						                       current_integratedOutput_ds, current_qya_d, current_qxa_d,
						                       current_prop_d, ay, ax, PsiProbeInit.get_dimj(), PsiProbeInit.get_dimi(),
						                       current_alphaInd_d, current_cufft_plan, current_stream);
						if (ax==0 & ay==0){
							cout <<"ON GPU" << endl;
							cout << "pars.stack.at(0,0,0,0) =  " << pars.stack.at(0,0,0,0) << endl;
							float s = 0;
							for (auto jj = 0; jj < pars.stack.get_dimj(); ++jj)s+=pars.stack.at(0,0,jj,0);
							cout << "sum = " << s << endl;
							if (std::isnan(s) | s < 0.01)
							{
								cout << "MESSED UP!!!\n";
								cout << "MESSED UP!!!\n";
								cout << "MESSED UP!!!\n";
								cout << "MESSED UP!!!\n";
								float ans;
								for (auto jj = 0; jj < pars.detectorAngles.size(); ++jj){
									hipMemcpy(&ans, current_integratedOutput_ds + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
									cout << "current_integratedOutput_ds = " << ans << endl;
								}
								for (auto jj = 0; jj < 100; ++jj){
									hipMemcpy(&ans, current_alphaInd_d + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
									cout << "current_alphaInd = " << ans << endl;
								}
								for (auto jj = 0; jj < 100; ++jj){
									hipMemcpy(&ans, current_qya_d + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
									cout << "qya_d = " << ans << endl;
								}
								for (auto jj = 0; jj < 100; ++jj){
									hipMemcpy(&ans, current_qxa_d + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
									cout << "qxa_d = " << ans << endl;
								}
								complex<float> deb;
								for (auto jj = 0; jj < 100; ++jj){
									hipMemcpy(&deb, current_psi_ds + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
									cout << "psi_ds = " << deb << endl;
								}
								for (auto jj = 0; jj < 100; ++jj){
									hipMemcpy(&deb, current_PsiProbeInit_d + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
									cout << "PsiProbeInit_d = " << deb << endl;
								}



							} //else{
//								float ans;
//								for (auto jj = 0; jj < pars.detectorAngles.size(); ++jj){
//									hipMemcpy(&ans, current_integratedOutput_ds + jj, sizeof(PRISM_FLOAT_PRECISION), hipMemcpyDeviceToHost);
//									cout << "current_integratedOutput_ds = " << ans << endl;
//								}
//							}
						}
						++Nstart;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));

			++stream_count;
		}


		// now launch CPU work
		PRISM_FFTW_INIT_THREADS();
		PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);
		if (pars.meta.also_do_CPU_work) {
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching CPU worker #" << t << '\n';
				// emplace_back is better whenever constructing a new object
				workers_CPU.emplace_back(thread([&pars, t, &trans, &alphaInd, &PsiProbeInit]() {
					size_t Nstart, Nstop, ay, ax;
					while (getWorkID(pars, Nstart, Nstop)) { // synchronously get work assignment
						while (Nstart != Nstop) {
							ay = Nstart / pars.xp.size();
							ax = Nstart % pars.xp.size();
							getMultisliceProbe_CPU(pars, trans, PsiProbeInit, ay, ax, alphaInd);
							++Nstart;
						}
					}

					cout << "CPU worker #" << t << " finished\n";
				}));
			}
		}
		// synchronize threads
		cout << "waiting on threads" << endl;
		for (auto& t:workers_GPU)t.join();

		for (auto& t:workers_CPU)t.join();
		PRISM_FFTW_CLEANUP_THREADS();
		cout << "threads done, cleaning up" << endl;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}

		// copy the results of the GPU, which are in pinned memory, back to the actual stack. The CPU work populates the
		// beginning, so make sure to copy from the offset of where the GPU started. Launch this copy on a background thread
		// while we cleanup the GPU
//		const size_t GPU_start_offset = (size_t)CPU_stop*pars.stack.get_dimk()*pars.stack.get_dimj()*pars.stack.get_dimi();
//		std::thread copy_t([&GPU_start_offset, &pars, &stack_ph](){
//			memcpy(&pars.stack[GPU_start_offset],
//			       &stack_ph[GPU_start_offset],
//			       (pars.stack.size()-GPU_start_offset) * sizeof(PRISM_FLOAT_PRECISION));
//		});

		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j){
			cudaErrchk(hipSetDevice(j));
//			cudaErrchk(hipDeviceSynchronize());
			cudaErrchk(hipFree(PsiProbeInit_d[j]));
			cudaErrchk(hipFree(trans_d[j]));
			cudaErrchk(hipFree(qxa_d[j]));
			cudaErrchk(hipFree(qya_d[j]));
			cudaErrchk(hipFree(prop_d[j]));
			cudaErrchk(hipFree(alphaInd_d[j]));
//			cudaErrchk(hipFree(integratedOutput_d[j]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipFree(psi_ds[s]));
			cudaErrchk(hipFree(psi_intensity_ds[s]));
			cudaErrchk(hipFree(integratedOutput_ds[s]));
			cufftErrchk(hipfftDestroy(cufft_plan[s]));
		}


		// free pinned memory
		cudaErrchk(hipHostFree(PsiProbeInit_ph));
		cudaErrchk(hipHostFree(trans_ph));
		cudaErrchk(hipHostFree(prop_ph));
		cudaErrchk(hipHostFree(qxa_ph));
		cudaErrchk(hipHostFree(qya_ph));
		cudaErrchk(hipHostFree(alphaInd_ph));
		cudaErrchk(hipHostFree(stack_ph));


		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamDestroy(streams[j]));
		}
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}

//		// make sure the copy is finished
//		copy_t.join();
	}
}
