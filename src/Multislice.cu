#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// PRISM is distributed under the GNU General Public License (GPL)
// If you use PRISM, we ask that you cite the following papers:

// / Calculate result of Multislice simulation using GPU and (potentially) CPU. Multiple GPU threads are launched, each with
// their own memory buffers. Page-locked host memory is allocated so that memory transfers to the GPU can occur asynchronously,
// and memory allocation for the GPU occurs only once, as each call to hipMalloc will potentially interrupt concurrent execution.
// Each GPU/CPU worker thread repeatedly calls getWorkID to be assigned probe positions to compute. This queue mechanism
// ensures that both the CPU and GPU are kept busy.

// For variable naming, the suffixes are "_d" for "device" (1 copy per GPU), "_ds" for "device stream (1 copy per stream), "_ph" for "pinned host"

#include "Multislice.cuh"
#include "Multislice.h"
#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"
#include "WorkDispatcher.h"
#include <iostream>
#include "fftw3.h"
#include "utility.h"
#include "utility.cuh"

namespace PRISM{
	extern std::mutex fftw_plan_lock;
	// computes the result of probe position ay,ax using the GPU. The effect of this function is the same as getMultisliceProbe_CPU
	__host__ void getMultisliceProbe_GPU_singlexfer(Parameters<PRISM_FLOAT_PRECISION>& pars,
	                                                PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                                PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                PRISM_FLOAT_PRECISION* output_ph,
	                                                PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                                PRISM_FLOAT_PRECISION* integratedOutput_ds,
									                const PRISM_FLOAT_PRECISION* qya_d,
									                const PRISM_FLOAT_PRECISION* qxa_d,
									                const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
									                const size_t ay,
									                const size_t ax,
									                const size_t dimj,
									                const size_t dimi,
									                const PRISM_FLOAT_PRECISION* alphaInd_d,
									                const hipfftHandle& plan,
									                hipStream_t& stream){

		// initialize psi
		PRISM_FLOAT_PRECISION yp = pars.yp[ay];
		PRISM_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t N = dimj*dimi;
		const size_t num_blocks = std::min(pars.target_num_blocks, (N - 1) / BLOCK_SIZE1D + 1);
		initializePsi<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, N, yp, xp);
		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, &trans_d[planeNum*N], N);
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, N);
			divide_inplace<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PRISM_MAKE_CU_COMPLEX(N, 0), N);
		}
		abs_squared<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_intensity_ds, psi_ds, N);
		formatOutput_GPU_integrate(pars, psi_intensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
}

	__host__ void getMultisliceProbe_GPU_singlexfer_batch(Parameters<PRISM_FLOAT_PRECISION>& pars,
	                                                      PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                                      PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                      PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                      PRISM_FLOAT_PRECISION* output_ph,
	                                                      PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                                      PRISM_FLOAT_PRECISION* integratedOutput_ds,
	                                                      const PRISM_FLOAT_PRECISION* qya_d,
	                                                      const PRISM_FLOAT_PRECISION* qxa_d,
	                                                      const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                                      const size_t Nstart,
	                                                      const size_t Nstop,
	                                                      const size_t dimj,
	                                                      const size_t dimi,
	                                                      const PRISM_FLOAT_PRECISION* alphaInd_d,
	                                                      const hipfftHandle& plan,
	                                                      hipStream_t& stream){
		const size_t N = dimj*dimi;
		const size_t num_blocks = std::min(pars.target_num_blocks, (N - 1) / BLOCK_SIZE1D + 1);
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();

			// initialize psi
			PRISM_FLOAT_PRECISION yp = pars.yp[ay];
			PRISM_FLOAT_PRECISION xp = pars.xp[ax];

//			initializePsi << < (N - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
			initializePsi << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
                (psi_ds + (batch_idx * N), PsiProbeInit_d, qya_d, qxa_d, N, yp, xp);
		}
		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
						(psi_ds + (batch_idx * N), &trans_d[planeNum * N], N);
			}
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
						(psi_ds + (batch_idx * N), prop_d, N);
				divide_inplace << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
						(psi_ds + (batch_idx * N), PRISM_MAKE_CU_COMPLEX(N, 0), N);
			}
		}
		abs_squared << < num_blocks, BLOCK_SIZE1D, 0, stream >> > (psi_intensity_ds, psi_ds, N*(Nstop-Nstart));
//		if (Nstart==0 ) {
//			cout << "after propagation" << endl;
//			{
////				std::complex<PRISM_FLOAT_PRECISION> ans_cx;
//				PRISM_FLOAT_PRECISION ans;
//				PRISM_FLOAT_PRECISION psi_intensity_sum = 0;
//				for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
//					psi_intensity_sum = 0;
//					cout << "BATCH ID = " << batch_idx << endl;
//					for (auto i = 0; i < 10; ++i) {
//						cudaErrchk(hipMemcpy(&ans, psi_intensity_ds + i + ((batch_idx * N)), sizeof(ans), hipMemcpyDeviceToHost));
//						cout << "psi_intensity_ds[" << i  + (batch_idx * N) << "] = " << ans << endl;
//					}
//					for (auto i = 0; i < pars.psiProbeInit.size(); ++i) {
//						cudaErrchk(hipMemcpy(&ans, psi_intensity_ds + i, sizeof(ans), hipMemcpyDeviceToHost));
//						psi_intensity_sum += ans;
//					}
//					Array2D<PRISM_FLOAT_PRECISION> debug = zeros_ND<2, PRISM_FLOAT_PRECISION >({{pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi()}});
//					cudaErrchk(hipMemcpy(&debug[0], psi_intensity_ds, pars.psiProbeInit.size() * sizeof(debug[0]), hipMemcpyDeviceToHost));
//					debug.toMRC_f("p1.mrc");
//					cudaErrchk(hipMemcpy(&debug[0], psi_intensity_ds + pars.psiProbeInit.size(), pars.psiProbeInit.size() * sizeof(debug[0]), hipMemcpyDeviceToHost));
//					debug.toMRC_f("p2.mrc");
//					cout << "PSI_INTENSITY SUM = " << psi_intensity_sum << endl;
//				}
//			}
//		}

		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();
			formatOutput_GPU_integrate(pars, psi_intensity_ds + (batch_idx * N),
			                           alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
		}
	}




	__host__ void getMultisliceProbe_GPU_streaming(Parameters<PRISM_FLOAT_PRECISION>& pars,
	                                               PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                               const complex<PRISM_FLOAT_PRECISION>* trans_ph,
	                                               PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                               PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
	                                               PRISM_FLOAT_PRECISION* output_ph,
	                                               PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                               PRISM_FLOAT_PRECISION* integratedOutput_ds,
	                                               const PRISM_FLOAT_PRECISION* qya_d,
	                                               const PRISM_FLOAT_PRECISION* qxa_d,
	                                               const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                               const size_t& ay,
	                                               const size_t& ax,
	                                               const size_t dimj,
	                                               const size_t dimi,
	                                               const PRISM_FLOAT_PRECISION* alphaInd_d,
	                                               const hipfftHandle& plan,
	                                               hipStream_t& stream){
		// initialize psi
		PRISM_FLOAT_PRECISION yp = pars.yp[ay];
		PRISM_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t N = dimj*dimi;
		const size_t num_blocks = std::min(pars.target_num_blocks, (N - 1) / BLOCK_SIZE1D + 1);
		initializePsi<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);


		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*N], N * sizeof(PRISM_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, trans_d, N);
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, N);
			divide_inplace<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PRISM_MAKE_CU_COMPLEX(N, 0), N);
		}
		abs_squared<<<num_blocks,BLOCK_SIZE1D, 0, stream>>>(psi_intensity_ds, psi_ds, N);
		formatOutput_GPU_integrate(pars, psi_intensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
	}


	__host__ void getMultisliceProbe_GPU_streaming_batch(Parameters<PRISM_FLOAT_PRECISION>& pars,
	                                                     PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                                     const complex<PRISM_FLOAT_PRECISION>* trans_ph,
	                                                     PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                     PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                     PRISM_FLOAT_PRECISION* output_ph,
	                                                     PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                                     PRISM_FLOAT_PRECISION* integratedOutput_ds,
	                                                     const PRISM_FLOAT_PRECISION* qya_d,
	                                                     const PRISM_FLOAT_PRECISION* qxa_d,
	                                                     const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                                     const size_t Nstart,
	                                                     const size_t Nstop,
	                                                     const size_t dimj,
	                                                     const size_t dimi,
	                                                     const PRISM_FLOAT_PRECISION* alphaInd_d,
	                                                     const hipfftHandle& plan,
	                                                     hipStream_t& stream){

		// initialize psi
		const size_t N = dimj*dimi;
		const size_t num_blocks = std::min(pars.target_num_blocks, (N - 1) / BLOCK_SIZE1D + 1);
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();
			PRISM_FLOAT_PRECISION yp = pars.yp[ay];
			PRISM_FLOAT_PRECISION xp = pars.xp[ax];
			initializePsi << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
			                                                (psi_ds + (batch_idx * N), PsiProbeInit_d, qya_d, qxa_d, N, yp, xp);
		}

		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {

			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*N], N * sizeof(PRISM_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
				                                                   (psi_ds + (batch_idx * N), trans_d, N);
			}
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
				multiply_inplace << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
				                                                   (psi_ds + (batch_idx * N), prop_d, N);
				divide_inplace << < num_blocks, BLOCK_SIZE1D, 0, stream >> >
				                                                 (psi_ds + (batch_idx * N), PRISM_MAKE_CU_COMPLEX(N, 0), N);
			}
		}
		abs_squared << < num_blocks, BLOCK_SIZE1D, 0, stream >> > (psi_intensity_ds, psi_ds, N*(Nstop-Nstart));
		for (auto batch_idx = 0; batch_idx < (Nstop-Nstart); ++batch_idx) {
			const size_t ay = (Nstart + batch_idx) / pars.xp.size();
			const size_t ax = (Nstart + batch_idx) % pars.xp.size();
			formatOutput_GPU_integrate(pars, psi_intensity_ds + (batch_idx * N),
			                           alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi, stream);
		}
	}

    __host__ void buildMultisliceOutput_GPU_singlexfer(Parameters <PRISM_FLOAT_PRECISION> &pars){
#ifdef PRISM_BUILDING_GUI
	    pars.progressbar->signalDescriptionMessage("Computing final output");
#endif
		// determine the batch size to use
	    pars.meta.batch_size_GPU = min(pars.meta.batch_size_target_GPU, pars.xp.size() * pars.yp.size() / (pars.meta.NUM_STREAMS_PER_GPU*pars.meta.NUM_GPUS));
		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU
		using namespace std;
		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
        hipStream_t *streams   = new hipStream_t[total_num_streams];
        hipfftHandle *cufft_plan = new hipfftHandle[total_num_streams];
//		hipStream_t streams[total_num_streams];
//		hipfftHandle cufft_plan[total_num_streams];


	    // batch parameters for cuFFT
	    const int rank = 2;
	    int n[] = {(int)pars.psiProbeInit.get_dimj(), (int)pars.psiProbeInit.get_dimi()};
	    const int howmany = pars.meta.batch_size_GPU;
	    cout <<"pars.meta.batch_size_GPU= " << pars.meta.batch_size_GPU<< endl;
	    int idist = n[0]*n[1];
	    int odist = n[0]*n[1];
	    int istride = 1;
	    int ostride = 1;
	    int *inembed = n;
	    int *onembed = n;

		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&streams[j]));
			cufftErrchk(hipfftPlanMany(&cufft_plan[j], rank, n, inembed, istride, idist, onembed, ostride, odist, PRISM_CUFFT_PLAN_TYPE, howmany));
//			cufftErrchk(hipfftPlan2d(&cufft_plan[j], pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(), PRISM_CUFFT_PLAN_TYPE));
			cufftErrchk(hipfftSetStream(cufft_plan[j], streams[j]));
		}


		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations


		// pointers to pinned host memory for async transfers
		std::complex<PRISM_FLOAT_PRECISION>  *PsiProbeInit_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *trans_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *prop_ph;
		PRISM_FLOAT_PRECISION                *qxa_ph;
		PRISM_FLOAT_PRECISION                *qya_ph;
		PRISM_FLOAT_PRECISION                *alphaInd_ph;
//		PRISM_FLOAT_PRECISION                *output_ph[total_num_streams];
		PRISM_FLOAT_PRECISION                **output_ph = new PRISM_FLOAT_PRECISION*[total_num_streams];
		// allocate pinned memory
		cudaErrchk(hipHostMalloc((void **)&PsiProbeInit_ph, pars.psiProbeInit.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&trans_ph,        pars.transmission.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&prop_ph,         pars.prop.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&qxa_ph,          pars.qxa.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&qya_ph,          pars.qya.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&alphaInd_ph,     pars.alphaInd.size()*sizeof(PRISM_FLOAT_PRECISION)));
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &output_ph[s], pars.output.get_dimi() * sizeof(PRISM_FLOAT_PRECISION)));
		}
		// copy host memory to pinned
		memcpy(PsiProbeInit_ph, &pars.psiProbeInit[0], pars.psiProbeInit.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(trans_ph,        &pars.transmission[0],        pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(prop_ph,         &pars.prop[0],    pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(qxa_ph,          &pars.qxa[0],     pars.qxa.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(qya_ph,          &pars.qya[0],     pars.qya.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(alphaInd_ph,     &pars.alphaInd[0],     pars.alphaInd.size() * sizeof(PRISM_FLOAT_PRECISION));


		// pointers to read-only GPU memory (one copy per GPU)
		PRISM_CUDA_COMPLEX_FLOAT **PsiProbeInit_d = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT **trans_d		  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT **prop_d 		  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qxa_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qya_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **alphaInd_d     = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];

		// pointers to read/write GPU memory (one per stream)
		PRISM_CUDA_COMPLEX_FLOAT **psi_ds 			   = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		PRISM_FLOAT_PRECISION    **psi_intensity_ds    = new PRISM_FLOAT_PRECISION*[total_num_streams];
		PRISM_FLOAT_PRECISION    **integratedOutput_ds = new PRISM_FLOAT_PRECISION*[total_num_streams];
//		PRISM_CUDA_COMPLEX_FLOAT *PsiProbeInit_d[pars.meta.NUM_GPUS];
//		PRISM_CUDA_COMPLEX_FLOAT *trans_d[pars.meta.NUM_GPUS];
//		PRISM_CUDA_COMPLEX_FLOAT *prop_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qxa_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qya_d[pars.meta.NUM_GPUS];
//	    PRISM_FLOAT_PRECISION    *alphaInd_d[pars.meta.NUM_GPUS];
//
//		// pointers to read/write GPU memory (one per stream)
//		PRISM_CUDA_COMPLEX_FLOAT *psi_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *psi_intensity_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *integratedOutput_ds[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &PsiProbeInit_d[g],     pars.psiProbeInit.size()   * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
			cudaErrchk(hipMalloc((void **) &trans_d[g],            pars.transmission.size()   * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
			cudaErrchk(hipMalloc((void **) &prop_d[g],             pars.prop.size()           * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &qxa_d[g],              pars.qxa.size()            * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &qya_d[g],              pars.qya.size()            * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &alphaInd_d[g],         pars.alphaInd.size()       * sizeof(PRISM_FLOAT_PRECISION)));
		}

	    cout << "pars.psiProbeInit.get_dimj()  = " << pars.psiProbeInit.get_dimj()<< endl;
	    cout << "pars.psiProbeInit.get_dimj() = " << pars.psiProbeInit.get_dimi()<< endl;
	    cout << "pars.psiProbeInit.size()  = " << pars.psiProbeInit.size()<< endl;
	    cout << "pars.meta.batch_size_GPU  = " << pars.meta.batch_size_GPU<< endl;
	    cout << "pars.meta.batch_size_GPU*pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]) = " << pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0]) << endl;
	    cout << "pars.meta.batch_size_GPU*pars.psiProbeInit.size() *  sizeof(PRISM_FLOAT_PRECISION) = " << pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)<< endl;

	    for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &psi_ds[s],              pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
			cudaErrchk(hipMalloc((void **) &psi_intensity_ds[s],    pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &integratedOutput_ds[s], pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(psi_ds[s], 0,                      pars.meta.batch_size_GPU*pars.psiProbeInit.size()  * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
			cudaErrchk(hipMemset(psi_intensity_ds[s], 0,            pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(integratedOutput_ds[s], 0,         pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
		}


		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(PsiProbeInit_d[g], &PsiProbeInit_ph[0],
			                      pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(trans_d[g], &trans_ph[0],
			                      pars.transmission.size() * sizeof(pars.transmission[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(prop_d[g], &prop_ph[0],
			                      pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(qxa_d[g], &qxa_ph[0],
			                      pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(qya_d[g], &qya_ph[0],
			                      pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(alphaInd_d[g], &alphaInd_ph[0],
			                      pars.alphaInd.size() * sizeof(pars.alphaInd[0]), hipMemcpyHostToDevice, streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceSynchronize());
		}

		size_t psi_size = pars.psiProbeInit.size();
		int stream_count = 0;
//		setWorkStartStop(0, pars.xp.size() * pars.yp.size());
	    const size_t PRISM_PRINT_FREQUENCY_PROBES = pars.xp.size() * pars.yp.size() / 10; // for printing status
		WorkDispatcher dispatcher(0, pars.xp.size() * pars.yp.size());
//		setWorkStartStop(0, 1);
		cout << " pars.xp.size()  = " << pars.xp.size()  << endl;
		cout << " pars.yp.size()  = " << pars.yp.size()  << endl;

		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipStream_t& current_stream = streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " on GPU #" << GPU_num << '\n';

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = PsiProbeInit_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_d = trans_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d  = prop_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qxa_d      = qxa_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qya_d      = qya_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_alphaInd_d = alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds           = psi_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_psi_intensity_ds    = psi_intensity_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_integratedOutput_ds = integratedOutput_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_output_ph           = output_ph[stream_count];
			hipfftHandle & current_cufft_plan = cufft_plan[stream_count];
			// launch a new thread
			workers_GPU.push_back(thread([&pars, current_trans_d, current_PsiProbeInit_d, current_alphaInd_d, &dispatcher,
					                                current_psi_ds, current_psi_intensity_ds, current_integratedOutput_ds,
					                                GPU_num, current_qya_d, current_qxa_d, current_output_ph, &current_cufft_plan,
					                                current_prop_d, &current_stream, &psi_size, stream_count, &PRISM_PRINT_FREQUENCY_PROBES]() {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU

#ifndef NDEBUG
				{
//					 check memory usage on the GPU
					std::lock_guard<mutex> lock(PRISM::mem_lock);
					size_t free_mem, total_mem;
					cudaErrchk(hipMemGetInfo(&free_mem, &total_mem));
					pars.max_mem = std::max(total_mem - free_mem, pars.max_mem);
//					cout << "max_mem = " << pars.max_mem << '\n';
				}
#endif // NDEBUG

				size_t Nstart, Nstop;
				Nstart=Nstop=0;
//				while (getWorkID(pars, Nstart, Nstop)){ // synchronously get work assignment
				while (dispatcher.getWork(Nstart, Nstop, pars.meta.batch_size_GPU)){ // synchronously get work assignment
					while (Nstart < Nstop){
						if (Nstart % PRISM_PRINT_FREQUENCY_PROBES < pars.meta.batch_size_GPU | Nstart == 100){
							cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << '\n';
						}
//						ay = Nstart / pars.xp.size();
//						ax = Nstart % pars.xp.size();
//						cout << "outside ax = " << ax << endl;
//						cout << "outside ay = " << ay <<vi  endl;
//							getMultisliceProbe_GPU_singlexfer(pars, current_trans_d, current_PsiProbeInit_d, current_psi_ds, current_output_ph,
//							                                  current_psi_intensity_ds,
//							                                  current_integratedOutput_ds, current_qya_d, current_qxa_d,
//							                                  current_prop_d, ay, ax, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
//							                                  current_alphaInd_d, current_cufft_plan, current_stream);
						getMultisliceProbe_GPU_singlexfer_batch(pars, current_trans_d, current_PsiProbeInit_d, current_psi_ds, current_output_ph,
						                                        current_psi_intensity_ds,
						                                        current_integratedOutput_ds, current_qya_d, current_qxa_d,
						                                        current_prop_d, Nstart, Nstop, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
						                                        current_alphaInd_d, current_cufft_plan, current_stream);
#ifdef PRISM_BUILDING_GUI
						pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
						Nstart=Nstop;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));

			++stream_count;
		}


		// now launch CPU work
		if (pars.meta.also_do_CPU_work){
			PRISM_FFTW_INIT_THREADS();
			PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations
			// If the batch size is too big, the work won't be spread over the threads, which will usually hurt more than the benefit
			// of batch FFT
			pars.meta.batch_size_CPU = min(pars.meta.batch_size_target_CPU, pars.xp.size() * pars.yp.size() / pars.meta.NUM_THREADS);
			cout << "multislice pars.meta.batch_size_CPU = " << pars.meta.batch_size_CPU << endl;
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching CPU worker #" << t << endl;
				// push_back is better whenever constructing a new object
				workers_CPU.push_back(thread([&pars, &dispatcher, t, &PRISM_PRINT_FREQUENCY_PROBES]() {
				size_t Nstart, Nstop, early_CPU_stop;
				Nstart=Nstop=0;
				// stop the CPU workers earlier than the GPU ones to prevent slower workers taking the last jobs and having to
				// wait longer for everything to complete
                    if (pars.meta.NUM_GPUS > 0){
                      // if there are no GPUs, make sure to do all work on CPU
                        early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0, pars.xp.size() * pars.yp.size() - pars.meta.gpu_cpu_ratio);
                    } else {
                        early_CPU_stop = pars.xp.size() * pars.yp.size();
                    }
					if (dispatcher.getWork(Nstart, Nstop, pars.meta.batch_size_CPU, early_CPU_stop)) { // synchronously get work assignment
                        Array1D<std::complex<PRISM_FLOAT_PRECISION> > psi_stack = zeros_ND<1, complex<PRISM_FLOAT_PRECISION> >({{pars.psiProbeInit.size() * pars.meta.batch_size_CPU}});

                        // setup batch FFTW parameters
                        const int rank = 2;
                        int n[] = {(int)pars.psiProbeInit.get_dimj(), (int)pars.psiProbeInit.get_dimi()};
                        const int howmany = pars.meta.batch_size_CPU;
                        int idist = n[0]*n[1];
                        int odist = n[0]*n[1];
                        int istride = 1;
                        int ostride = 1;
                        int *inembed = n;
                        int *onembed = n;
                        unique_lock<mutex> gatekeeper(fftw_plan_lock);
                        PRISM_FFTW_PLAN plan_forward = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
                                                                                 reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
                                                                                 istride, idist,
                                                                                 reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
                                                                                 ostride, odist,
                                                                                 FFTW_FORWARD, FFTW_MEASURE);
                        PRISM_FFTW_PLAN plan_inverse = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
                                                                                 reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
                                                                                 istride, idist,
                                                                                 reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
                                                                                 ostride, odist,
                                                                                 FFTW_BACKWARD, FFTW_MEASURE);

                        gatekeeper.unlock();
						do {
							//	cout << "Nstop = " << Nstop << endl;
							while (Nstart < Nstop) {
                                if (Nstart % PRISM_PRINT_FREQUENCY_PROBES  < pars.meta.batch_size_CPU | Nstart == 100){
                                    cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << endl;
                                }
//							ay = Nstart / pars.xp.size();
//							ax = Nstart % pars.xp.size();
//                            if (ay==7){
//                                cout << "ax = " << ax << endl;
//								cout << "ay = " << ay << endl;
//                            }
//							getMultisliceProbe_CPU(pars, ay, ax, plan_forward, plan_inverse, psi);
                                getMultisliceProbe_CPU_batch(pars, Nstart, Nstop, plan_forward, plan_inverse, psi_stack);
#ifdef PRISM_BUILDING_GUI
                                pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
                                //++Nstart;
                                Nstart=Nstop;
							}
							if (Nstop >= early_CPU_stop) break;
						} while(dispatcher.getWork(Nstart, Nstop, pars.meta.batch_size_CPU, early_CPU_stop));
						gatekeeper.lock();
						PRISM_FFTW_DESTROY_PLAN(plan_forward);
						PRISM_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
//					cout << "CPU worker #" << t << " finished\n";
			
					}));
				
			}
			cout << "Waiting on CPU threads..." << endl;
			for (auto& t:workers_CPU)t.join();
			PRISM_FFTW_CLEANUP_THREADS();
		}
		// synchronize threads
		cout << "Waiting on GPU threads..." << endl;
		for (auto& t:workers_GPU)t.join();



		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}

		// copy the results of the GPU, which are in pinned memory, back to the actual stack. The CPU work populates the
		// beginning, so make sure to copy from the offset of where the GPU started. Launch this copy on a background thread
		// while we cleanup the GPU
//		const size_t GPU_start_offset = (size_t)CPU_stop*pars.output.get_dimk()*pars.output.get_dimj()*pars.output.get_dimi();
//		std::thread copy_t([&GPU_start_offset, &pars, &stack_ph](){
//			memcpy(&pars.output[GPU_start_offset],
//			       &stack_ph[GPU_start_offset],
//			       (pars.output.size()-GPU_start_offset) * sizeof(PRISM_FLOAT_PRECISION));
//		});

		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j){
			cudaErrchk(hipSetDevice(j));
//			cudaErrchk(hipDeviceSynchronize());
			cudaErrchk(hipFree(PsiProbeInit_d[j]));
			cudaErrchk(hipFree(trans_d[j]));
			cudaErrchk(hipFree(qxa_d[j]));
			cudaErrchk(hipFree(qya_d[j]));
			cudaErrchk(hipFree(prop_d[j]));
			cudaErrchk(hipFree(alphaInd_d[j]));
//			cudaErrchk(hipFree(integratedOutput_d[j]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipFree(psi_ds[s]));
			cudaErrchk(hipFree(psi_intensity_ds[s]));
			cudaErrchk(hipFree(integratedOutput_ds[s]));
			cufftErrchk(hipfftDestroy(cufft_plan[s]));
		}


		// free pinned memory
		cudaErrchk(hipHostFree(PsiProbeInit_ph));
		cudaErrchk(hipHostFree(trans_ph));
		cudaErrchk(hipHostFree(prop_ph));
		cudaErrchk(hipHostFree(qxa_ph));
		cudaErrchk(hipHostFree(qya_ph));
		cudaErrchk(hipHostFree(alphaInd_ph));
		for (auto s =0; s < total_num_streams; ++s){
			cudaErrchk(hipHostFree(output_ph[s]));
		}

		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamDestroy(streams[j]));
		}
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}
		delete[] streams;
		delete[] cufft_plan;
		delete[] PsiProbeInit_d;
		delete[] trans_d;
		delete[] prop_d;
		delete[] qxa_d;
		delete[] qya_d;
		delete[] alphaInd_d;
		delete[] psi_ds;
		delete[] psi_intensity_ds;
		delete[] integratedOutput_ds;
		delete[] output_ph;
	}




	__host__ void buildMultisliceOutput_GPU_streaming(Parameters <PRISM_FLOAT_PRECISION> &pars){
#ifdef PRISM_BUILDING_GUI
		pars.progressbar->signalDescriptionMessage("Computing final output");
#endif

		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		using namespace std;

		// determine batch size
		pars.meta.batch_size_GPU = min(pars.meta.batch_size_target_GPU, pars.xp.size() * pars.yp.size() / (pars.meta.NUM_STREAMS_PER_GPU*pars.meta.NUM_GPUS));
		cout << "multislice pars.meta.batch_size_CPU = " << pars.meta.batch_size_CPU << endl;
		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
        hipStream_t *streams   = new hipStream_t[total_num_streams];
        hipfftHandle *cufft_plan = new hipfftHandle[total_num_streams];
		cout <<"total_num_streams = " << total_num_streams<< endl;
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&streams[j]));
			cufftErrchk(hipfftPlan2d(&cufft_plan[j], pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(), PRISM_CUFFT_PLAN_TYPE));
			cufftErrchk(hipfftSetStream(cufft_plan[j], streams[j]));
		}


		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations


		// pointers to pinned host memory for async transfers
		std::complex<PRISM_FLOAT_PRECISION>  *PsiProbeInit_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *trans_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *prop_ph;
		PRISM_FLOAT_PRECISION                *qxa_ph;
		PRISM_FLOAT_PRECISION                *qya_ph;
		PRISM_FLOAT_PRECISION                *alphaInd_ph;
//		PRISM_FLOAT_PRECISION                *output_ph[total_num_streams];
		PRISM_FLOAT_PRECISION                **output_ph = new PRISM_FLOAT_PRECISION*[total_num_streams];
		// allocate pinned memory
		cudaErrchk(hipHostMalloc((void **)&PsiProbeInit_ph, pars.psiProbeInit.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&trans_ph,        pars.transmission.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&prop_ph,         pars.prop.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&qxa_ph,          pars.qxa.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&qya_ph,          pars.qya.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&alphaInd_ph,     pars.alphaInd.size()*sizeof(PRISM_FLOAT_PRECISION)));
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &output_ph[s], pars.output.get_dimi() * sizeof(PRISM_FLOAT_PRECISION)));
		}
		// copy host memory to pinned
		memcpy(PsiProbeInit_ph, &pars.psiProbeInit[0], pars.psiProbeInit.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(trans_ph,        &pars.transmission[0],        pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(prop_ph,         &pars.prop[0],    pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(qxa_ph,          &pars.qxa[0],     pars.qxa.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(qya_ph,          &pars.qya[0],     pars.qya.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(alphaInd_ph,     &pars.alphaInd[0],     pars.alphaInd.size() * sizeof(PRISM_FLOAT_PRECISION));


		// pointers to read-only GPU memory (one copy per GPU)
		PRISM_CUDA_COMPLEX_FLOAT **PsiProbeInit_d = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT **prop_d 	   	  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qxa_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qya_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **alphaInd_d 	  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];

		// pointers to read/write GPU memory (one per stream)
		PRISM_CUDA_COMPLEX_FLOAT **trans_ds 		   = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		PRISM_CUDA_COMPLEX_FLOAT **psi_ds  		       = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		PRISM_FLOAT_PRECISION    **psi_intensity_ds    = new PRISM_FLOAT_PRECISION*[total_num_streams];
		PRISM_FLOAT_PRECISION    **integratedOutput_ds = new PRISM_FLOAT_PRECISION*[total_num_streams];
//		// pointers to read-only GPU memory (one copy per GPU)
//		PRISM_CUDA_COMPLEX_FLOAT *PsiProbeInit_d[pars.meta.NUM_GPUS];
//		PRISM_CUDA_COMPLEX_FLOAT *prop_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qxa_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qya_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *alphaInd_d[pars.meta.NUM_GPUS];
//
//		// pointers to read/write GPU memory (one per stream)
//		PRISM_CUDA_COMPLEX_FLOAT *trans_ds[total_num_streams];
//		PRISM_CUDA_COMPLEX_FLOAT *psi_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *psi_intensity_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *integratedOutput_ds[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &PsiProbeInit_d[g],     pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &prop_d[g],             pars.prop.size()           * sizeof(pars.prop[0])));
			cudaErrchk(hipMalloc((void **) &qxa_d[g],              pars.qxa.size()            * sizeof(pars.qxa[0])));
			cudaErrchk(hipMalloc((void **) &qya_d[g],              pars.qya.size()            * sizeof(pars.qya[0])));
			cudaErrchk(hipMalloc((void **) &alphaInd_d[g],         pars.alphaInd.size()            * sizeof(pars.alphaInd[0])));
		}

		cout << "pars.psiProbeInit.size()  = " << pars.psiProbeInit.size()<< endl;
		cout << "pars.meta.batch_size_GPU  = " << pars.meta.batch_size_GPU<< endl;
		cout << "pars.meta.batch_size_GPU*pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]) = " << pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0]) << endl;
		cout << "pars.meta.batch_size_GPU*pars.psiProbeInit.size() *  sizeof(PRISM_FLOAT_PRECISION) = " << pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)<< endl;
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &trans_ds[s],            pars.transmission.get_dimj() * pars.transmission.get_dimi() * sizeof(pars.transmission[0])));
			cudaErrchk(hipMalloc((void **) &psi_ds[s],              pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &psi_intensity_ds[s],    pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &integratedOutput_ds[s], pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(psi_ds[s], 0, pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMemset(psi_intensity_ds[s], 0, pars.meta.batch_size_GPU*pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(integratedOutput_ds[s], 0, pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
		}


		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(PsiProbeInit_d[g], &PsiProbeInit_ph[0],
			                           pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
//			cudaErrchk(hipMemcpyAsync(trans_d[g], &trans_ph[0],
//			                           trans.size() * sizeof(trans[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(prop_d[g], &prop_ph[0],
			                           pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(qxa_d[g], &qxa_ph[0],
			                           pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(qya_d[g], &qya_ph[0],
			                           pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(alphaInd_d[g], &alphaInd_ph[0],
			                           pars.alphaInd.size() * sizeof(pars.alphaInd[0]), hipMemcpyHostToDevice, streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceSynchronize());
		}

		size_t psi_size = pars.psiProbeInit.size();
		int stream_count = 0;
//		setWorkStartStop(0, pars.xp.size() * pars.yp.size());
		const size_t PRISM_PRINT_FREQUENCY_PROBES = pars.xp.size() * pars.yp.size() / 10; // for printing status
		WorkDispatcher dispatcher(0, pars.xp.size() * pars.yp.size());
		// If the batch size is too big, the work won't be spread over the threads, which will usually hurt more than the benefit
		// of batch FFT

		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipStream_t& current_stream = streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << endl;

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = PsiProbeInit_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d   = prop_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qxa_d       = qxa_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qya_d       = qya_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_alphaInd_d  = alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_ds         = trans_ds[stream_count];
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds           = psi_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_psi_intensity_ds    = psi_intensity_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_integratedOutput_ds = integratedOutput_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_output_ph           = output_ph[stream_count];
			hipfftHandle & current_cufft_plan                   = cufft_plan[stream_count];
			// launch a new thread
			// push_back is better whenever constructing a new object
			workers_GPU.push_back(thread([&pars, current_trans_ds, trans_ph, current_PsiProbeInit_d, current_alphaInd_d, &dispatcher,
					                                current_psi_ds, current_psi_intensity_ds, current_integratedOutput_ds,
					                                GPU_num, current_qya_d, current_qxa_d, current_output_ph, current_cufft_plan,
					                                current_prop_d, &current_stream, &psi_size, stream_count, &PRISM_PRINT_FREQUENCY_PROBES]()  {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU


#ifndef NDEBUG
				{
//					 check memory usage on the GPU
					std::lock_guard<mutex> lock(PRISM::mem_lock);
					size_t free_mem, total_mem;
					cudaErrchk(hipMemGetInfo(&free_mem, &total_mem));
					pars.max_mem = std::max(total_mem - free_mem, pars.max_mem);
//					cout << "max_mem = " << pars.max_mem << endl;
				}
#endif // NDEBUG

				size_t Nstart, Nstop;
				Nstart=Nstop=0;
//				while (getWorkID(pars, Nstart, Nstop)){ // synchronously get work assignment
				while (dispatcher.getWork(Nstart, Nstop, pars.meta.batch_size_GPU)){ // synchronously get work assignment
					while (Nstart < Nstop){
						if (Nstart % PRISM_PRINT_FREQUENCY_PROBES < pars.meta.batch_size_GPU | Nstart == 100){
							cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << endl;
						}
//						ay = Nstart / pars.xp.size();
//						ax = Nstart % pars.xp.size();
//
//						getMultisliceProbe_GPU_streaming(pars, current_trans_ds, trans_ph, current_PsiProbeInit_d, current_psi_ds,
//						                                 current_output_ph, current_psi_intensity_ds,
//						                                 current_integratedOutput_ds, current_qya_d, current_qxa_d,
//						                                 current_prop_d, ay, ax, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
//						                                 current_alphaInd_d, current_cufft_plan, current_stream);
						getMultisliceProbe_GPU_streaming_batch(pars, current_trans_ds, trans_ph, current_PsiProbeInit_d, current_psi_ds,
						                                       current_output_ph, current_psi_intensity_ds,
						                                       current_integratedOutput_ds, current_qya_d, current_qxa_d,
						                                       current_prop_d, Nstart, Nstop, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
						                                       current_alphaInd_d, current_cufft_plan, current_stream);
#ifdef PRISM_BUILDING_GUI
						pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
//						++Nstart;
						Nstart = Nstop;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));

			++stream_count;
		}


		// now launch CPU work

		if (pars.meta.also_do_CPU_work){
			PRISM_FFTW_INIT_THREADS();
			PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching CPU worker #" << t << endl;
				// push_back is better whenever constructing a new object
				workers_CPU.push_back(thread([&pars, &dispatcher, t, &PRISM_PRINT_FREQUENCY_PROBES]() {
				size_t Nstart, Nstop, early_CPU_stop;
				Nstart=Nstop=0;
				// stop the CPU workers earlier than the GPU ones to prevent slower workers taking the last jobs and having to
				// wait longer for everything to complete
                if (pars.meta.NUM_GPUS > 0){
                        // if there are no GPUs, make sure to do all work on CPU
                            early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0, pars.xp.size() * pars.yp.size() - pars.meta.gpu_cpu_ratio);
                } else {
                            early_CPU_stop = pars.xp.size() * pars.yp.size();
                }
				if (dispatcher.getWork(Nstart, Nstop, pars.meta.batch_size_CPU, early_CPU_stop)) { // synchronously get work assignment
					Array1D<std::complex<PRISM_FLOAT_PRECISION> > psi_stack = zeros_ND<1, complex<PRISM_FLOAT_PRECISION> >({{pars.psiProbeInit.size() * pars.meta.batch_size_CPU}});

					// setup batch FFTW parameters
					const int rank = 2;
					int n[] = {(int)pars.psiProbeInit.get_dimj(), (int)pars.psiProbeInit.get_dimi()};
					const int howmany = pars.meta.batch_size_CPU;
					int idist = n[0]*n[1];
					int odist = n[0]*n[1];
					int istride = 1;
					int ostride = 1;
					int *inembed = n;
					int *onembed = n;
					unique_lock<mutex> gatekeeper(fftw_plan_lock);
					PRISM_FFTW_PLAN plan_forward = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
					                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
					                                                         istride, idist,
					                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
					                                                         ostride, odist,
					                                                         FFTW_FORWARD, FFTW_MEASURE);
					PRISM_FFTW_PLAN plan_inverse = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
					                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
					                                                         istride, idist,
					                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
					                                                         ostride, odist,
					                                                         FFTW_BACKWARD, FFTW_MEASURE);

					gatekeeper.unlock();
					do {
						//	cout << "Nstop = " << Nstop << endl;
						while (Nstart < Nstop) {
							if (Nstart % PRISM_PRINT_FREQUENCY_PROBES  < pars.meta.batch_size_CPU | Nstart == 100){
								cout << "Computing Probe Position #" << Nstart << "/" << pars.xp.size() * pars.yp.size() << endl;
							}
//							ay = Nstart / pars.xp.size();
//							ax = Nstart % pars.xp.size();
//                            if (ay==7){
//                                cout << "ax = " << ax << endl;
//								cout << "ay = " << ay << endl;
//                            }
//							getMultisliceProbe_CPU(pars, ay, ax, plan_forward, plan_inverse, psi);
							getMultisliceProbe_CPU_batch(pars, Nstart, Nstop, plan_forward, plan_inverse, psi_stack);
#ifdef PRISM_BUILDING_GUI
							pars.progressbar->signalOutputUpdate(Nstart, pars.xp.size() * pars.yp.size());
#endif
							//++Nstart;
							Nstart=Nstop;
						}
						if (Nstop >= early_CPU_stop) break;
					} while(dispatcher.getWork(Nstart, Nstop, pars.meta.batch_size_CPU, early_CPU_stop));
					gatekeeper.lock();
					PRISM_FFTW_DESTROY_PLAN(plan_forward);
					PRISM_FFTW_DESTROY_PLAN(plan_inverse);
					gatekeeper.unlock();
				}
				cout << "CPU worker #" << t << " finished\n";

			}));

			}
			cout << "Waiting on GPU threads..." << endl;
			for (auto& t:workers_CPU)t.join();
			PRISM_FFTW_CLEANUP_THREADS();
		}
		// synchronize threads
		cout << "Waiting on GPU threads..." << endl;
		for (auto& t:workers_GPU)t.join();



		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}

		// copy the results of the GPU, which are in pinned memory, back to the actual stack. The CPU work populates the
		// beginning, so make sure to copy from the offset of where the GPU started. Launch this copy on a background thread
		// while we cleanup the GPU
//		const size_t GPU_start_offset = (size_t)CPU_stop*pars.output.get_dimk()*pars.output.get_dimj()*pars.output.get_dimi();
//		std::thread copy_t([&GPU_start_offset, &pars, &stack_ph](){
//			memcpy(&pars.output[GPU_start_offset],
//			       &stack_ph[GPU_start_offset],
//			       (pars.output.size()-GPU_start_offset) * sizeof(PRISM_FLOAT_PRECISION));
//		});

		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j){
			cudaErrchk(hipSetDevice(j));
//			cudaErrchk(hipDeviceSynchronize());
			cudaErrchk(hipFree(PsiProbeInit_d[j]));
			cudaErrchk(hipFree(trans_ds[j]));
			cudaErrchk(hipFree(qxa_d[j]));
			cudaErrchk(hipFree(qya_d[j]));
			cudaErrchk(hipFree(prop_d[j]));
			cudaErrchk(hipFree(alphaInd_d[j]));
//			cudaErrchk(hipFree(integratedOutput_d[j]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipFree(psi_ds[s]));
			cudaErrchk(hipFree(psi_intensity_ds[s]));
			cudaErrchk(hipFree(integratedOutput_ds[s]));
			cufftErrchk(hipfftDestroy(cufft_plan[s]));
		}


		// free pinned memory
		cudaErrchk(hipHostFree(PsiProbeInit_ph));
		cudaErrchk(hipHostFree(trans_ph));
		cudaErrchk(hipHostFree(prop_ph));
		cudaErrchk(hipHostFree(qxa_ph));
		cudaErrchk(hipHostFree(qya_ph));
		cudaErrchk(hipHostFree(alphaInd_ph));
		for (auto s =0; s < total_num_streams; ++s){
			cudaErrchk(hipHostFree(output_ph[s]));
		}

		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamDestroy(streams[j]));
		}
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}
		delete[] streams;
		delete[] cufft_plan;
		delete[] PsiProbeInit_d;
		delete[] trans_ds;
		delete[] prop_d;
		delete[] qxa_d;
		delete[] qya_d;
		delete[] alphaInd_d;
		delete[] psi_ds;
		delete[] psi_intensity_ds;
		delete[] integratedOutput_ds;
		delete[] output_ph;
	}

}
