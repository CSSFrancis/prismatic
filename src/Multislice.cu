#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// PRISM is distributed under the GNU General Public License (GPL)
// If you use PRISM, we ask that you cite the following papers:

// / Calculate result of Multislice simulation using GPU and (potentially) CPU. Multiple GPU threads are launched, each with
// their own memory buffers. Page-locked host memory is allocated so that memory transfers to the GPU can occur asynchronously,
// and memory allocation for the GPU occurs only once, as each call to hipMalloc will potentially interrupt concurrent execution.
// Each GPU/CPU worker thread repeatedly calls getWorkID to be assigned probe positions to compute. This queue mechanism
// ensures that both the CPU and GPU are kept busy.

// For variable naming, the suffixes are "_d" for "device" (1 copy per GPU), "_ds" for "device stream (1 copy per stream), "_ph" for "pinned host"

#include "Multislice.cuh"
#include "Multislice.h"
#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"
#include "WorkDispatcher.h"
#include <iostream>
#include "fftw3.h"
#include "utility.h"
#include "utility.cuh"

namespace PRISM{
	extern std::mutex fftw_plan_lock;
	// computes the result of probe position ay,ax using the GPU. The effect of this function is the same as getMultisliceProbe_CPU
	__host__ void getMultisliceProbe_GPU_singlexfer(Parameters<PRISM_FLOAT_PRECISION>& pars,
	                                                PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                                PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                                PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
	                                                PRISM_FLOAT_PRECISION* output_ph,
	                                                PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                                PRISM_FLOAT_PRECISION* integratedOutput_ds,
									                const PRISM_FLOAT_PRECISION* qya_d,
									                const PRISM_FLOAT_PRECISION* qxa_d,
									                const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
									                const size_t& ay,
									                const size_t& ax,
									                const size_t dimj,
									                const size_t dimi,
									                const PRISM_FLOAT_PRECISION* alphaInd_d,
									                const hipfftHandle& plan,
									                hipStream_t& stream){
		/*
		if (ay==0 && ax == 0) {
			cout << "dimj = " << dimj << endl;
			cout << "dimi = " << dimi << endl;
			{
				std::complex<PRISM_FLOAT_PRECISION> ans_cx;
				PRISM_FLOAT_PRECISION ans;
//				for (auto i = 0; i < 10; ++i) {
//					cudaErrchk(hipMemcpy(&ans_cx, trans_d + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
//					cout << "trans_d[" << i << "] = " << ans_cx << endl;
//				}

//				for (auto i = 0; i < 10; ++i) {
//					cudaErrchk(hipMemcpy(&ans_cx, PsiProbeInit_d + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
//					cout << "PsiProbeInit_d[" << i << "] = " << ans_cx << endl;
//				}
//
//				for (auto i = 0; i < 10; ++i) {
//					cudaErrchk(hipMemcpy(&ans_cx, psi_ds + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
//					cout << "psi_ds[" << i << "] = " << ans_cx << endl;
//				}
//
//				for (auto i = 0; i < 10; ++i) {
//					cudaErrchk(hipMemcpy(&ans, alphaInd_d + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
//					cout << "alphaInd_d[" << i << "] = " << ans << endl;
//				}
				for (auto i = 0; i < 10; ++i) {
					cudaErrchk(hipMemcpy(&ans_cx, prop_d + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
					cout << "prop_d[" << i << "] = " << ans_cx << endl;
				}

				for (auto i = 0; i < 10; ++i) {
					cudaErrchk(hipMemcpy(&ans, qya_d + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
					cout << "qya_d[" << i << "] = " << ans << endl;
				}
				for (auto i = 0; i < 10; ++i) {
					cudaErrchk(hipMemcpy(&ans, qxa_d + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
					cout << "qxa_d[" << i << "] = " << ans << endl;
				}

			}
		}

		*/

		// initialize psi
		PRISM_FLOAT_PRECISION yp = pars.yp[ay];
		PRISM_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t N = dimj*dimi;
		initializePsi<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);

		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			/*
			if (ax == 0 && ay == 0) {
				cout << " planeNum = " << planeNum << endl;
				std::complex<PRISM_FLOAT_PRECISION> ans_cx;
				PRISM_FLOAT_PRECISION ans;
				for (auto i = 0; i < 10; ++i) {
					cudaErrchk(hipMemcpy(&ans_cx, psi_ds + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
					cout << "end psi_ds[" << i << "] = " << ans_cx << endl;
				}
			}
			 */
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, &trans_d[planeNum*N], N);
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, N);
			divide_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PRISM_MAKE_CU_COMPLEX(N, 0), N);
		}
		//cout << "pars.numPlanes = " << pars.numPlanes << endl;


		abs_squared<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_intensity_ds, psi_ds, N);
		/*
		if (ax == 0 && ay == 0){
			std::complex<PRISM_FLOAT_PRECISION> ans_cx;
			PRISM_FLOAT_PRECISION ans;
			for (auto i = 0; i < 10; ++i) {
				cudaErrchk(hipMemcpy(&ans_cx, psi_ds + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
				cout << "end psi_ds[" << i << "] = " << ans_cx << endl;
			}

			for (auto i = 0; i < 10; ++i) {
				cudaErrchk(hipMemcpy(&ans, psi_intensity_ds + i, sizeof(ans_cx), hipMemcpyDeviceToHost));
				cout << "end psi_intensity_ds[" << i << "] = " << ans << endl;
			}
		}
		*/
		formatOutput_GPU_integrate(pars, psi_intensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
}


	__host__ void getMultisliceProbe_GPU_streaming(Parameters<PRISM_FLOAT_PRECISION>& pars,
	                                               PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                               const complex<PRISM_FLOAT_PRECISION>* trans_ph,
	                                               PRISM_CUDA_COMPLEX_FLOAT* PsiProbeInit_d,
	                                               PRISM_CUDA_COMPLEX_FLOAT* psi_ds,
	                                               PRISM_FLOAT_PRECISION* output_ph,
	                                               PRISM_FLOAT_PRECISION* psi_intensity_ds,
	                                               PRISM_FLOAT_PRECISION* integratedOutput_ds,
	                                               const PRISM_FLOAT_PRECISION* qya_d,
	                                               const PRISM_FLOAT_PRECISION* qxa_d,
	                                               const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                               const size_t& ay,
	                                               const size_t& ax,
	                                               const size_t dimj,
	                                               const size_t dimi,
	                                               const PRISM_FLOAT_PRECISION* alphaInd_d,
	                                               const hipfftHandle& plan,
	                                               hipStream_t& stream){

		// initialize psi
		PRISM_FLOAT_PRECISION yp = pars.yp[ay];
		PRISM_FLOAT_PRECISION xp = pars.xp[ax];
		const size_t N = dimj*dimi;
		initializePsi<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PsiProbeInit_d, qya_d, qxa_d, dimj*dimi, yp, xp);

		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*N], N * sizeof(PRISM_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_BACKWARD));
			multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, trans_d, N);
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_ds[0], &psi_ds[0], HIPFFT_FORWARD));
			multiply_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, prop_d, N);
			divide_inplace<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_ds, PRISM_MAKE_CU_COMPLEX(N, 0), N);
		}
		abs_squared<<<(N-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_intensity_ds, psi_ds, N);
		formatOutput_GPU_integrate(pars, psi_intensity_ds, alphaInd_d, output_ph, integratedOutput_ds, ay, ax, dimj, dimi,stream);
	}

    __host__ void buildMultisliceOutput_GPU_singlexfer(Parameters <PRISM_FLOAT_PRECISION> &pars){

		cout << "pars.numPlanes = " << pars.numPlanes << endl;
		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU
		using namespace std;

		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
        hipStream_t *streams   = new hipStream_t[total_num_streams];
        hipfftHandle *cufft_plan = new hipfftHandle[total_num_streams];
//		hipStream_t streams[total_num_streams];
//		hipfftHandle cufft_plan[total_num_streams];


		cout <<"total_num_streams = " << total_num_streams<< endl;
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&streams[j]));
			cufftErrchk(hipfftPlan2d(&cufft_plan[j], pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(), PRISM_CUFFT_PLAN_TYPE));
			cufftErrchk(hipfftSetStream(cufft_plan[j], streams[j]));
		}


		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations


		// pointers to pinned host memory for async transfers
		std::complex<PRISM_FLOAT_PRECISION>  *PsiProbeInit_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *trans_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *prop_ph;
		PRISM_FLOAT_PRECISION                *qxa_ph;
		PRISM_FLOAT_PRECISION                *qya_ph;
		PRISM_FLOAT_PRECISION                *alphaInd_ph;
//		PRISM_FLOAT_PRECISION                *output_ph[total_num_streams];
		PRISM_FLOAT_PRECISION                **output_ph = new PRISM_FLOAT_PRECISION*[total_num_streams];
		// allocate pinned memory
		cudaErrchk(hipHostMalloc((void **)&PsiProbeInit_ph, pars.psiProbeInit.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&trans_ph,        pars.transmission.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&prop_ph,         pars.prop.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&qxa_ph,          pars.qxa.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&qya_ph,          pars.qya.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&alphaInd_ph,     pars.alphaInd.size()*sizeof(PRISM_FLOAT_PRECISION)));
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &output_ph[s], pars.output.get_dimj() * pars.output.get_dimi() * sizeof(PRISM_FLOAT_PRECISION)));
		}
		// copy host memory to pinned
		memcpy(PsiProbeInit_ph, &pars.psiProbeInit[0], pars.psiProbeInit.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(trans_ph,        &pars.transmission[0],        pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(prop_ph,         &pars.prop[0],    pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(qxa_ph,          &pars.qxa[0],     pars.qxa.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(qya_ph,          &pars.qya[0],     pars.qya.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(alphaInd_ph,     &pars.alphaInd[0],     pars.alphaInd.size() * sizeof(PRISM_FLOAT_PRECISION));


		// pointers to read-only GPU memory (one copy per GPU)
		PRISM_CUDA_COMPLEX_FLOAT **PsiProbeInit_d = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT **trans_d		  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT **prop_d 		  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qxa_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qya_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **alphaInd_d     = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];

		// pointers to read/write GPU memory (one per stream)
		PRISM_CUDA_COMPLEX_FLOAT **psi_ds 			   = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		PRISM_FLOAT_PRECISION    **psi_intensity_ds    = new PRISM_FLOAT_PRECISION*[total_num_streams];
		PRISM_FLOAT_PRECISION    **integratedOutput_ds = new PRISM_FLOAT_PRECISION*[total_num_streams];
//		PRISM_CUDA_COMPLEX_FLOAT *PsiProbeInit_d[pars.meta.NUM_GPUS];
//		PRISM_CUDA_COMPLEX_FLOAT *trans_d[pars.meta.NUM_GPUS];
//		PRISM_CUDA_COMPLEX_FLOAT *prop_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qxa_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qya_d[pars.meta.NUM_GPUS];
//	    PRISM_FLOAT_PRECISION    *alphaInd_d[pars.meta.NUM_GPUS];
//
//		// pointers to read/write GPU memory (one per stream)
//		PRISM_CUDA_COMPLEX_FLOAT *psi_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *psi_intensity_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *integratedOutput_ds[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &PsiProbeInit_d[g],     pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &trans_d[g],            pars.transmission.size()               * sizeof(pars.transmission[0])));
			cudaErrchk(hipMalloc((void **) &prop_d[g],             pars.prop.size()           * sizeof(pars.prop[0])));
			cudaErrchk(hipMalloc((void **) &qxa_d[g],              pars.qxa.size()            * sizeof(pars.qxa[0])));
			cudaErrchk(hipMalloc((void **) &qya_d[g],              pars.qya.size()            * sizeof(pars.qya[0])));
			cudaErrchk(hipMalloc((void **) &alphaInd_d[g],         pars.alphaInd.size()            * sizeof(pars.alphaInd[0])));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &psi_ds[s],              pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &psi_intensity_ds[s],    pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &integratedOutput_ds[s], pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(psi_ds[s], 0, pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMemset(psi_intensity_ds[s], 0, pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(integratedOutput_ds[s], 0, pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
		}


		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(PsiProbeInit_d[g], &PsiProbeInit_ph[0],
			                      pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(trans_d[g], &trans_ph[0],
			                      pars.transmission.size() * sizeof(pars.transmission[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(prop_d[g], &prop_ph[0],
			                      pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(qxa_d[g], &qxa_ph[0],
			                      pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(qya_d[g], &qya_ph[0],
			                      pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(alphaInd_d[g], &alphaInd_ph[0],
			                      pars.alphaInd.size() * sizeof(pars.alphaInd[0]), hipMemcpyHostToDevice, streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceSynchronize());
		}

		size_t psi_size = pars.psiProbeInit.size();
		int stream_count = 0;
//		setWorkStartStop(0, pars.xp.size() * pars.yp.size());
		WorkDispatcher dispatcher(0, pars.xp.size() * pars.yp.size(), 1);
//		setWorkStartStop(0, 1);
		cout << " pars.xp.size()  = " << pars.xp.size()  << endl;
		cout << " pars.yp.size()  = " << pars.yp.size()  << endl;

		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipStream_t& current_stream = streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << '\n';

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = PsiProbeInit_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_d = trans_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d  = prop_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qxa_d      = qxa_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qya_d      = qya_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_alphaInd_d = alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds           = psi_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_psi_intensity_ds    = psi_intensity_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_integratedOutput_ds = integratedOutput_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_output_ph           = output_ph[stream_count];
			hipfftHandle & current_cufft_plan = cufft_plan[stream_count];
			// launch a new thread
			workers_GPU.push_back(thread([&pars, current_trans_d, current_PsiProbeInit_d, current_alphaInd_d, &dispatcher,
					                                current_psi_ds, current_psi_intensity_ds, current_integratedOutput_ds,
					                                GPU_num, current_qya_d, current_qxa_d, current_output_ph, &current_cufft_plan,
					                                current_prop_d, &current_stream, &psi_size, stream_count]() {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU
				size_t Nstart, Nstop, ay, ax;
				Nstart=Nstop=0;
//				while (getWorkID(pars, Nstart, Nstop)){ // synchronously get work assignment
				while (dispatcher.getWork(Nstart, Nstop)){ // synchronously get work assignment
					while (Nstart != Nstop){
						ay = Nstart / pars.xp.size();
						ax = Nstart % pars.xp.size();
//						cout << "outside ax = " << ax << endl;
//						cout << "outside ay = " << ay << endl;
							getMultisliceProbe_GPU_singlexfer(pars, current_trans_d, current_PsiProbeInit_d, current_psi_ds, current_output_ph,
							                                  current_psi_intensity_ds,
							                                  current_integratedOutput_ds, current_qya_d, current_qxa_d,
							                                  current_prop_d, ay, ax, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
							                                  current_alphaInd_d, current_cufft_plan, current_stream);
						++Nstart;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));

			++stream_count;
		}


		// now launch CPU work

		if (pars.meta.also_do_CPU_work){
			PRISM_FFTW_INIT_THREADS();
			PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching CPU worker #" << t << '\n';
				// push_back is better whenever constructing a new object
				workers_CPU.push_back(thread([&pars, &dispatcher, t]() {
				size_t Nstart, Nstop, early_CPU_stop, ay, ax;
				Nstart=Nstop=0;
				// stop the CPU workers earlier than the GPU ones to prevent slower workers taking the last jobs and having to
				// wait longer for everything to complete
                                if (pars.meta.NUM_GPUS > 0){
                                      // if there are no GPUs, make sure to do all work on CPU
                                        early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0, pars.xp.size() * pars.yp.size() - pars.meta.gpu_cpu_ratio);
                                } else {
                                        early_CPU_stop = pars.xp.size() * pars.yp.size();
                                }
					if (dispatcher.getWork(Nstart, Nstop, early_CPU_stop)) { // synchronously get work assignment
						Array2D<complex<PRISM_FLOAT_PRECISION> > psi(pars.psiProbeInit);
						unique_lock<mutex> gatekeeper(fftw_plan_lock);
						PRISM_FFTW_PLAN plan_forward = PRISM_FFTW_PLAN_DFT_2D(psi.get_dimj(), psi.get_dimi(),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  FFTW_FORWARD, FFTW_MEASURE);
						PRISM_FFTW_PLAN plan_inverse = PRISM_FFTW_PLAN_DFT_2D(psi.get_dimj(), psi.get_dimi(),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  FFTW_BACKWARD, FFTW_MEASURE);
						gatekeeper.unlock();
						do {
							//	cout << "Nstop = " << Nstop << endl;
							while (Nstart != Nstop) {
								ay = Nstart / pars.xp.size();
								ax = Nstart % pars.xp.size();
//                            if (ay==7){
//                                cout << "ax = " << ax << endl;
//								cout << "ay = " << ay << endl;
//                            }
								getMultisliceProbe_CPU(pars, ay, ax, plan_forward, plan_inverse, psi);
								++Nstart;
							}
							if (Nstop >= early_CPU_stop) break;
						} while(dispatcher.getWork(Nstart, Nstop, early_CPU_stop));
						gatekeeper.lock();
						PRISM_FFTW_DESTROY_PLAN(plan_forward);
						PRISM_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
					cout << "CPU worker #" << t << " finished\n";
			
					}));
				
			}
			cout << "Waiting on GPU threads..." << endl;
			for (auto& t:workers_CPU)t.join();
			PRISM_FFTW_CLEANUP_THREADS();
		}
		// synchronize threads
		cout << "Waiting on GPU threads..." << endl;
		for (auto& t:workers_GPU)t.join();



		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}

		// copy the results of the GPU, which are in pinned memory, back to the actual stack. The CPU work populates the
		// beginning, so make sure to copy from the offset of where the GPU started. Launch this copy on a background thread
		// while we cleanup the GPU
//		const size_t GPU_start_offset = (size_t)CPU_stop*pars.output.get_dimk()*pars.output.get_dimj()*pars.output.get_dimi();
//		std::thread copy_t([&GPU_start_offset, &pars, &stack_ph](){
//			memcpy(&pars.output[GPU_start_offset],
//			       &stack_ph[GPU_start_offset],
//			       (pars.output.size()-GPU_start_offset) * sizeof(PRISM_FLOAT_PRECISION));
//		});

		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j){
			cudaErrchk(hipSetDevice(j));
//			cudaErrchk(hipDeviceSynchronize());
			cudaErrchk(hipFree(PsiProbeInit_d[j]));
			cudaErrchk(hipFree(trans_d[j]));
			cudaErrchk(hipFree(qxa_d[j]));
			cudaErrchk(hipFree(qya_d[j]));
			cudaErrchk(hipFree(prop_d[j]));
			cudaErrchk(hipFree(alphaInd_d[j]));
//			cudaErrchk(hipFree(integratedOutput_d[j]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipFree(psi_ds[s]));
			cudaErrchk(hipFree(psi_intensity_ds[s]));
			cudaErrchk(hipFree(integratedOutput_ds[s]));
			cufftErrchk(hipfftDestroy(cufft_plan[s]));
		}


		// free pinned memory
		cudaErrchk(hipHostFree(PsiProbeInit_ph));
		cudaErrchk(hipHostFree(trans_ph));
		cudaErrchk(hipHostFree(prop_ph));
		cudaErrchk(hipHostFree(qxa_ph));
		cudaErrchk(hipHostFree(qya_ph));
		cudaErrchk(hipHostFree(alphaInd_ph));
		for (auto s =0; s < total_num_streams; ++s){
			cudaErrchk(hipHostFree(output_ph[s]));
		}

		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamDestroy(streams[j]));
		}
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}
		delete[] streams;
		delete[] cufft_plan;
		delete[] PsiProbeInit_d;
		delete[] trans_d;
		delete[] prop_d;
		delete[] qxa_d;
		delete[] qya_d;
		delete[] alphaInd_d;
		delete[] psi_ds;
		delete[] psi_intensity_ds;
		delete[] integratedOutput_ds;
		delete[] output_ph;
	}




	__host__ void buildMultisliceOutput_GPU_streaming(Parameters <PRISM_FLOAT_PRECISION> &pars){


		// populate the Multislice output stack dividing the work between GPUs and CPU cores.
		// this version assumes the full trans array fits into DRAM on each GPU

		using namespace std;

		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
        hipStream_t *streams   = new hipStream_t[total_num_streams];
        hipfftHandle *cufft_plan = new hipfftHandle[total_num_streams];
		cout <<"total_num_streams = " << total_num_streams<< endl;
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&streams[j]));
			cufftErrchk(hipfftPlan2d(&cufft_plan[j], pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(), PRISM_CUFFT_PLAN_TYPE));
			cufftErrchk(hipfftSetStream(cufft_plan[j], streams[j]));
		}


		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations


		// pointers to pinned host memory for async transfers
		std::complex<PRISM_FLOAT_PRECISION>  *PsiProbeInit_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *trans_ph;
		std::complex<PRISM_FLOAT_PRECISION>  *prop_ph;
		PRISM_FLOAT_PRECISION                *qxa_ph;
		PRISM_FLOAT_PRECISION                *qya_ph;
		PRISM_FLOAT_PRECISION                *alphaInd_ph;
//		PRISM_FLOAT_PRECISION                *output_ph[total_num_streams];
		PRISM_FLOAT_PRECISION                **output_ph = new PRISM_FLOAT_PRECISION*[total_num_streams];
		// allocate pinned memory
		cudaErrchk(hipHostMalloc((void **)&PsiProbeInit_ph, pars.psiProbeInit.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&trans_ph,        pars.transmission.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&prop_ph,         pars.prop.size()*sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **)&qxa_ph,          pars.qxa.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&qya_ph,          pars.qya.size()*sizeof(PRISM_FLOAT_PRECISION)));
		cudaErrchk(hipHostMalloc((void **)&alphaInd_ph,     pars.alphaInd.size()*sizeof(PRISM_FLOAT_PRECISION)));
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &output_ph[s], pars.output.get_dimj() * pars.output.get_dimi() * sizeof(PRISM_FLOAT_PRECISION)));
		}
		// copy host memory to pinned
		memcpy(PsiProbeInit_ph, &pars.psiProbeInit[0], pars.psiProbeInit.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(trans_ph,        &pars.transmission[0],        pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(prop_ph,         &pars.prop[0],    pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(qxa_ph,          &pars.qxa[0],     pars.qxa.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(qya_ph,          &pars.qya[0],     pars.qya.size() * sizeof(PRISM_FLOAT_PRECISION));
		memcpy(alphaInd_ph,     &pars.alphaInd[0],     pars.alphaInd.size() * sizeof(PRISM_FLOAT_PRECISION));


		// pointers to read-only GPU memory (one copy per GPU)
		PRISM_CUDA_COMPLEX_FLOAT **PsiProbeInit_d = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_CUDA_COMPLEX_FLOAT **prop_d 	   	  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qxa_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **qya_d 		  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];
		PRISM_FLOAT_PRECISION    **alphaInd_d 	  = new PRISM_FLOAT_PRECISION*[pars.meta.NUM_GPUS];

		// pointers to read/write GPU memory (one per stream)
		PRISM_CUDA_COMPLEX_FLOAT **trans_ds 		   = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		PRISM_CUDA_COMPLEX_FLOAT **psi_ds  		       = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		PRISM_FLOAT_PRECISION    **psi_intensity_ds    = new PRISM_FLOAT_PRECISION*[total_num_streams];
		PRISM_FLOAT_PRECISION    **integratedOutput_ds = new PRISM_FLOAT_PRECISION*[total_num_streams];
//		// pointers to read-only GPU memory (one copy per GPU)
//		PRISM_CUDA_COMPLEX_FLOAT *PsiProbeInit_d[pars.meta.NUM_GPUS];
//		PRISM_CUDA_COMPLEX_FLOAT *prop_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qxa_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *qya_d[pars.meta.NUM_GPUS];
//		PRISM_FLOAT_PRECISION    *alphaInd_d[pars.meta.NUM_GPUS];
//
//		// pointers to read/write GPU memory (one per stream)
//		PRISM_CUDA_COMPLEX_FLOAT *trans_ds[total_num_streams];
//		PRISM_CUDA_COMPLEX_FLOAT *psi_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *psi_intensity_ds[total_num_streams];
//		PRISM_FLOAT_PRECISION    *integratedOutput_ds[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &PsiProbeInit_d[g],     pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &prop_d[g],             pars.prop.size()           * sizeof(pars.prop[0])));
			cudaErrchk(hipMalloc((void **) &qxa_d[g],              pars.qxa.size()            * sizeof(pars.qxa[0])));
			cudaErrchk(hipMalloc((void **) &qya_d[g],              pars.qya.size()            * sizeof(pars.qya[0])));
			cudaErrchk(hipMalloc((void **) &alphaInd_d[g],         pars.alphaInd.size()            * sizeof(pars.alphaInd[0])));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &trans_ds[s],            pars.transmission.size()               * sizeof(pars.transmission[0])));
			cudaErrchk(hipMalloc((void **) &psi_ds[s],              pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMalloc((void **) &psi_intensity_ds[s],    pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMalloc((void **) &integratedOutput_ds[s], pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(psi_ds[s], 0, pars.psiProbeInit.size()        * sizeof(pars.psiProbeInit[0])));
			cudaErrchk(hipMemset(psi_intensity_ds[s], 0, pars.psiProbeInit.size()        * sizeof(PRISM_FLOAT_PRECISION)));
			cudaErrchk(hipMemset(integratedOutput_ds[s], 0, pars.detectorAngles.size() * sizeof(PRISM_FLOAT_PRECISION)));
		}


		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(PsiProbeInit_d[g], &PsiProbeInit_ph[0],
			                           pars.psiProbeInit.size() * sizeof(pars.psiProbeInit[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
//			cudaErrchk(hipMemcpyAsync(trans_d[g], &trans_ph[0],
//			                           trans.size() * sizeof(trans[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(prop_d[g], &prop_ph[0],
			                           pars.prop.size() * sizeof(pars.prop[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(qxa_d[g], &qxa_ph[0],
			                           pars.qxa.size() * sizeof(pars.qxa[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(qya_d[g], &qya_ph[0],
			                           pars.qya.size() * sizeof(pars.qya[0]), hipMemcpyHostToDevice, streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cout << "stream_id = " << stream_id << endl;
			cudaErrchk(hipMemcpyAsync(alphaInd_d[g], &alphaInd_ph[0],
			                           pars.alphaInd.size() * sizeof(pars.alphaInd[0]), hipMemcpyHostToDevice, streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceSynchronize());
		}

		size_t psi_size = pars.psiProbeInit.size();
		int stream_count = 0;
//		setWorkStartStop(0, pars.xp.size() * pars.yp.size());
		WorkDispatcher dispatcher(0, pars.xp.size() * pars.yp.size(), 1);
		for (auto t = 0; t < total_num_streams; ++t){
			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipStream_t& current_stream = streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << '\n';

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_PsiProbeInit_d = PsiProbeInit_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d   = prop_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qxa_d       = qxa_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_qya_d       = qya_d[GPU_num];
			PRISM_FLOAT_PRECISION *current_alphaInd_d  = alphaInd_d[GPU_num];

			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_ds         = trans_ds[stream_count];
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds           = psi_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_psi_intensity_ds    = psi_intensity_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_integratedOutput_ds = integratedOutput_ds[stream_count];
			PRISM_FLOAT_PRECISION *current_output_ph           = output_ph[stream_count];
			hipfftHandle & current_cufft_plan                   = cufft_plan[stream_count];
			// launch a new thread
			// push_back is better whenever constructing a new object
			workers_GPU.push_back(thread([&pars, current_trans_ds, trans_ph, current_PsiProbeInit_d, current_alphaInd_d, &dispatcher,
					                                current_psi_ds, current_psi_intensity_ds, current_integratedOutput_ds,
					                                GPU_num, current_qya_d, current_qxa_d, current_output_ph, current_cufft_plan,
					                                current_prop_d, &current_stream, &psi_size, stream_count]() {

				// set the GPU context
				cudaErrchk(hipSetDevice(GPU_num)); // set current GPU
				size_t Nstart, Nstop, ay, ax;
				Nstart=Nstop=0;
//				while (getWorkID(pars, Nstart, Nstop)){ // synchronously get work assignment
				while (dispatcher.getWork(Nstart, Nstop)){ // synchronously get work assignment
					while (Nstart != Nstop){
						ay = Nstart / pars.xp.size();
						ax = Nstart % pars.xp.size();

						getMultisliceProbe_GPU_streaming(pars, current_trans_ds, trans_ph, current_PsiProbeInit_d, current_psi_ds,
						                                 current_output_ph, current_psi_intensity_ds,
						                                 current_integratedOutput_ds, current_qya_d, current_qxa_d,
						                                 current_prop_d, ay, ax, pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi(),
						                                 current_alphaInd_d, current_cufft_plan, current_stream);

						++Nstart;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));

			++stream_count;
		}


		// now launch CPU work

		if (pars.meta.also_do_CPU_work){
			PRISM_FFTW_INIT_THREADS();
			PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching CPU worker #" << t << '\n';
				// push_back is better whenever constructing a new object
				workers_CPU.push_back(thread([&pars, &dispatcher, t]() {
				size_t Nstart, Nstop, early_CPU_stop, ay, ax;
				Nstart=Nstop=0;
				// stop the CPU workers earlier than the GPU ones to prevent slower workers taking the last jobs and having to
				// wait longer for everything to complete
                                if (pars.meta.NUM_GPUS > 0){
                                      // if there are no GPUs, make sure to do all work on CPU
                                        early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0, pars.xp.size() * pars.yp.size() - pars.meta.gpu_cpu_ratio);
                                } else {
                                        early_CPU_stop = pars.xp.size() * pars.yp.size();
                                }
					if (dispatcher.getWork(Nstart, Nstop, early_CPU_stop)) { // synchronously get work assignment
						Array2D<complex<PRISM_FLOAT_PRECISION> > psi(pars.psiProbeInit);
						unique_lock<mutex> gatekeeper(fftw_plan_lock);
						PRISM_FFTW_PLAN plan_forward = PRISM_FFTW_PLAN_DFT_2D(psi.get_dimj(), psi.get_dimi(),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  FFTW_FORWARD, FFTW_MEASURE);
						PRISM_FFTW_PLAN plan_inverse = PRISM_FFTW_PLAN_DFT_2D(psi.get_dimj(), psi.get_dimi(),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi[0]),
																			  FFTW_BACKWARD, FFTW_MEASURE);
						gatekeeper.unlock();
						do {
							//	cout << "Nstop = " << Nstop << endl;
							while (Nstart != Nstop) {
								ay = Nstart / pars.xp.size();
								ax = Nstart % pars.xp.size();
//                            if (ay==7){
//                                cout << "ax = " << ax << endl;
//								cout << "ay = " << ay << endl;
//                            }
								getMultisliceProbe_CPU(pars, ay, ax, plan_forward, plan_inverse, psi);
								++Nstart;
							}
							if (Nstop >= early_CPU_stop) break;
						} while(dispatcher.getWork(Nstart, Nstop, early_CPU_stop));
						gatekeeper.lock();
						PRISM_FFTW_DESTROY_PLAN(plan_forward);
						PRISM_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
					cout << "CPU worker #" << t << " finished\n";

				}));

			}
			cout << "Waiting on GPU threads..." << endl;
			for (auto& t:workers_CPU)t.join();
			PRISM_FFTW_CLEANUP_THREADS();
		}
		// synchronize threads
		cout << "Waiting on GPU threads..." << endl;
		for (auto& t:workers_GPU)t.join();



		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			hipSetDevice(g);
			hipDeviceSynchronize();
		}

		// copy the results of the GPU, which are in pinned memory, back to the actual stack. The CPU work populates the
		// beginning, so make sure to copy from the offset of where the GPU started. Launch this copy on a background thread
		// while we cleanup the GPU
//		const size_t GPU_start_offset = (size_t)CPU_stop*pars.output.get_dimk()*pars.output.get_dimj()*pars.output.get_dimi();
//		std::thread copy_t([&GPU_start_offset, &pars, &stack_ph](){
//			memcpy(&pars.output[GPU_start_offset],
//			       &stack_ph[GPU_start_offset],
//			       (pars.output.size()-GPU_start_offset) * sizeof(PRISM_FLOAT_PRECISION));
//		});

		// synchronize GPUs and cleanup data
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j){
			cudaErrchk(hipSetDevice(j));
//			cudaErrchk(hipDeviceSynchronize());
			cudaErrchk(hipFree(PsiProbeInit_d[j]));
			cudaErrchk(hipFree(trans_ds[j]));
			cudaErrchk(hipFree(qxa_d[j]));
			cudaErrchk(hipFree(qya_d[j]));
			cudaErrchk(hipFree(prop_d[j]));
			cudaErrchk(hipFree(alphaInd_d[j]));
//			cudaErrchk(hipFree(integratedOutput_d[j]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipFree(psi_ds[s]));
			cudaErrchk(hipFree(psi_intensity_ds[s]));
			cudaErrchk(hipFree(integratedOutput_ds[s]));
			cufftErrchk(hipfftDestroy(cufft_plan[s]));
		}


		// free pinned memory
		cudaErrchk(hipHostFree(PsiProbeInit_ph));
		cudaErrchk(hipHostFree(trans_ph));
		cudaErrchk(hipHostFree(prop_ph));
		cudaErrchk(hipHostFree(qxa_ph));
		cudaErrchk(hipHostFree(qya_ph));
		cudaErrchk(hipHostFree(alphaInd_ph));
		for (auto s =0; s < total_num_streams; ++s){
			cudaErrchk(hipHostFree(output_ph[s]));
		}

		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamDestroy(streams[j]));
		}
		for (auto j = 0; j < pars.meta.NUM_GPUS; ++j) {
			cudaErrchk(hipSetDevice(j));
			cudaErrchk(hipDeviceReset());
		}
		delete[] streams;
		delete[] cufft_plan;
		delete[] PsiProbeInit_d;
		delete[] trans_ds;
		delete[] prop_d;
		delete[] qxa_d;
		delete[] qya_d;
		delete[] alphaInd_d;
		delete[] psi_ds;
		delete[] psi_intensity_ds;
		delete[] integratedOutput_ds;
		delete[] output_ph;
	}

}
