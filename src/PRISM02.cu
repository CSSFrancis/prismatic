#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// PRISM is distributed under the GNU General Public License (GPL)
// If you use PRISM, we ask that you cite the following papers:

// For variable naming, the suffixes are "_d" for "device" (1 copy per GPU), "_ds" for "device stream (1 copy per stream), "_ph" for "pinned host"

#include "PRISM02.cuh"
#include "PRISM02.h"
#include <thread>
#include "WorkDispatcher.h"
#include "hipfft/hipfft.h"
#include "utility.cuh"
#include "params.cuh"


namespace PRISM {
	using namespace std;
	inline void createStreamsAndPlans2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                  CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		// create CUDA streams
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		cuda_pars.streams 		    = new hipStream_t[total_num_streams];
		cuda_pars.cufft_plans		= new hipfftHandle[total_num_streams];
		cuda_pars.cufft_plans_small = new hipfftHandle[total_num_streams];

		// batch parameters for cuFFT
		const int rank      = 2;
		int n[]             = {(int)pars.imageSize[0], (int)pars.imageSize[1]};
		const int howmany   = pars.meta.batch_size_GPU;
		int idist           = n[0]*n[1];
		int odist           = n[0]*n[1];
		int istride         = 1;
		int ostride         = 1;
		int *inembed        = n;
		int *onembed        = n;

		int n_small[]       = {(int)pars.qyInd.size(), (int)pars.qxInd.size()};
		int idist_small     = n_small[0]*n_small[1];
		int odist_small     = n_small[0]*n_small[1];
		int *inembed_small  = n_small;
		int *onembed_small  = n_small;

		// create cuFFT plans and CUDA streams
		for (auto j = 0; j < total_num_streams; ++j) {
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamCreate(&cuda_pars.streams[j]));
			cufftErrchk(hipfftPlanMany(&cuda_pars.cufft_plans[j], rank, n, inembed, istride, idist, onembed, ostride, odist, PRISM_CUFFT_PLAN_TYPE, howmany));
			cufftErrchk(hipfftPlanMany(&cuda_pars.cufft_plans_small[j], rank, n_small, inembed_small, istride, idist_small, onembed_small, ostride, odist_small, PRISM_CUFFT_PLAN_TYPE, howmany));
			cufftErrchk(hipfftSetStream(cuda_pars.cufft_plans[j], cuda_pars.streams[j]));
			cufftErrchk(hipfftSetStream(cuda_pars.cufft_plans_small[j], cuda_pars.streams[j]));
		}
	}

	inline void allocatePinnedHostMemory_singlexfer2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                          CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;

		// allocate pinned memory
		cuda_pars.Scompact_slice_ph = new std::complex<PRISM_FLOAT_PRECISION>*[total_num_streams];
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &cuda_pars.Scompact_slice_ph[s],
			                          pars.Scompact.get_dimj() * pars.Scompact.get_dimi() *
			                          sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		}
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.trans_ph,      pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.prop_ph,       pars.prop.size()         * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.qxInd_ph,      pars.qxInd.size()        * sizeof(size_t)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.qyInd_ph,      pars.qyInd.size()        * sizeof(size_t)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.beamsIndex_ph, pars.beamsIndex.size()   * sizeof(size_t)));

	}

	inline void allocatePinnedHostMemory_streaming2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                         CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;

		// allocate pinned memory
		cuda_pars.Scompact_slice_ph = new std::complex<PRISM_FLOAT_PRECISION>*[total_num_streams];
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostMalloc((void **) &cuda_pars.Scompact_slice_ph[s],
			                          pars.Scompact.get_dimj() * pars.Scompact.get_dimi() *
			                          sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		}
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.trans_ph,      pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.prop_ph,       pars.prop.size()         * sizeof(std::complex<PRISM_FLOAT_PRECISION>)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.qxInd_ph,      pars.qxInd.size()        * sizeof(size_t)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.qyInd_ph,      pars.qyInd.size()        * sizeof(size_t)));
		cudaErrchk(hipHostMalloc((void **) &cuda_pars.beamsIndex_ph, pars.beamsIndex.size()   * sizeof(size_t)));
	}

	inline void copyToPinnedMemory_singlexfer2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                          CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;

		// copy host memory to pinned
		for (auto s = 0; s < total_num_streams; ++s) {
			memset(cuda_pars.Scompact_slice_ph[s], 0, pars.Scompact.get_dimj() * pars.Scompact.get_dimi() *
			                                          sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		}
		memcpy(cuda_pars.trans_ph,      &pars.transmission[0], pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(cuda_pars.prop_ph,       &pars.prop[0],         pars.prop.size()         * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(cuda_pars.qxInd_ph,      &pars.qxInd[0],        pars.qxInd.size()        * sizeof(size_t));
		memcpy(cuda_pars.qyInd_ph,      &pars.qyInd[0],        pars.qyInd.size()        * sizeof(size_t));
		memcpy(cuda_pars.beamsIndex_ph, &pars.beamsIndex[0],   pars.beamsIndex.size()   * sizeof(size_t));
	}

	inline void copyToPinnedMemory_streaming2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                         CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;

		// copy host memory to pinned
		for (auto s = 0; s < total_num_streams; ++s) {
			memset(cuda_pars.Scompact_slice_ph[s], 0, pars.Scompact.get_dimj() * pars.Scompact.get_dimi() *
			                                          sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		}
		memcpy(cuda_pars.trans_ph,      &pars.transmission[0], pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(cuda_pars.prop_ph,       &pars.prop[0],         pars.prop.size()         * sizeof(std::complex<PRISM_FLOAT_PRECISION>));
		memcpy(cuda_pars.qxInd_ph,      &pars.qxInd[0],        pars.qxInd.size()        * sizeof(size_t));
		memcpy(cuda_pars.qyInd_ph,      &pars.qyInd[0],        pars.qyInd.size()        * sizeof(size_t));
		memcpy(cuda_pars.beamsIndex_ph, &pars.beamsIndex[0],   pars.beamsIndex.size()   * sizeof(size_t));
	}

	inline void allocateDeviceMemory_singlexfer2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                            CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;

		// allocate memory on the device

		// pointers to read-only GPU memory (one copy per GPU)
		cuda_pars.trans_d       = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		cuda_pars.prop_d        = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		cuda_pars.qxInd_d       = new size_t*[pars.meta.NUM_GPUS];
		cuda_pars.qyInd_d       = new size_t*[pars.meta.NUM_GPUS];
		cuda_pars.beamsIndex_d  = new size_t*[pars.meta.NUM_GPUS];

//		// pointers to read/write GPU memory (one per stream)
		cuda_pars.psi_ds       = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		cuda_pars.psi_small_ds = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMalloc((void **) &cuda_pars.trans_d[g],      pars.transmission.size() * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.prop_d[g],       pars.prop.size()         * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qxInd_d[g],      pars.qxInd.size()        * sizeof(size_t)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qyInd_d[g],      pars.qyInd.size()        * sizeof(size_t)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.beamsIndex_d[g], pars.beamsIndex.size()   * sizeof(size_t)));
		}

		// allocate memory per stream and 0 it
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psi_ds[s],
			                      pars.meta.batch_size_GPU*pars.imageSize[0] * pars.imageSize[1] * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psi_small_ds[s],
			                      pars.meta.batch_size_GPU*pars.qxInd.size() * pars.qyInd.size() * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMemset(cuda_pars.psi_ds[s], 0,
			                      pars.meta.batch_size_GPU*pars.imageSize[0] * pars.imageSize[1] * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMemset(cuda_pars.psi_small_ds[s], 0,
			                      pars.meta.batch_size_GPU*pars.qxInd.size() * pars.qyInd.size() * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
		}
	}

	inline void allocateDeviceMemory_streaming2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                           CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		// pointers to read-only GPU memory (one copy per GPU)
		cuda_pars.prop_d  = new PRISM_CUDA_COMPLEX_FLOAT*[pars.meta.NUM_GPUS];
		cuda_pars.qxInd_d                   = new size_t*[pars.meta.NUM_GPUS];
		cuda_pars.qyInd_d                   = new size_t*[pars.meta.NUM_GPUS];
		cuda_pars.beamsIndex_d              = new size_t*[pars.meta.NUM_GPUS];

		// pointers to read/write GPU memory (one per stream)
		cuda_pars.psi_ds       = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		cuda_pars.psi_small_ds = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];
		cuda_pars.trans_d    = new PRISM_CUDA_COMPLEX_FLOAT*[total_num_streams];

		// allocate memory on each GPU
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));

			cudaErrchk(hipMalloc((void **) &cuda_pars.prop_d[g], pars.prop.size() * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qxInd_d[g], pars.qxInd.size() * sizeof(size_t)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.qyInd_d[g], pars.qyInd.size() * sizeof(size_t)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.beamsIndex_d[g], pars.beamsIndex.size() * sizeof(size_t)));
		}

		// allocate memory per stream and 0 it
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipMalloc((void **) &cuda_pars.trans_d[s],
			                      pars.imageSize[0] * pars.imageSize[1] * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psi_ds[s],
			                      pars.meta.batch_size_GPU*pars.imageSize[0] * pars.imageSize[1] * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMalloc((void **) &cuda_pars.psi_small_ds[s],
			                      pars.meta.batch_size_GPU*pars.qxInd.size() * pars.qyInd.size() * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMemset(cuda_pars.psi_ds[s], 0,
			                      pars.meta.batch_size_GPU*pars.imageSize[0] * pars.imageSize[1] * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
			cudaErrchk(hipMemset(cuda_pars.psi_small_ds[s], 0,
			                      pars.meta.batch_size_GPU*pars.qxInd.size() * pars.qyInd.size() * sizeof(PRISM_CUDA_COMPLEX_FLOAT)));
		}
	}

	inline void copyToDeviceMemory_singlexfer2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                          CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipMemcpyAsync(cuda_pars.trans_d[g], &cuda_pars.trans_ph[0],
			                           pars.transmission.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>),
			                           hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));

			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.prop_d[g], &cuda_pars.prop_ph[0],
			                           pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>),
			                           hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qxInd_d[g], &cuda_pars.qxInd_ph[0],
			                           pars.qxInd.size() * sizeof(size_t), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qyInd_d[g], &cuda_pars.qyInd_ph[0],
			                           pars.qyInd.size() * sizeof(size_t), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.beamsIndex_d[g], &cuda_pars.beamsIndex_ph[0],
			                           pars.beamsIndex.size() * sizeof(size_t), hipMemcpyHostToDevice,
			                           cuda_pars.streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			hipSetDevice(g);
			hipDeviceSynchronize();
		}
	}

	inline void copyToDeviceMemory_streaming2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                         CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		// Copy memory to each GPU asynchronously from the pinned host memory spaces.
		// The streams are laid out so that consecutive streams represent different GPUs. If we
		// have more than one stream per GPU, then we want to interleave as much as possible
		int stream_id = 0;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			stream_id = g;
			cudaErrchk(hipSetDevice(g));
//			cudaErrchk(hipMemcpyAsync(trans_ds[g], &trans_ph[0],
//			                           pars.imageSize[0] * pars.imageSize[1] * sizeof(std::complex<PRISM_FLOAT_PRECISION>),
//			                           hipMemcpyHostToDevice, streams[stream_id]));

			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.prop_d[g], &cuda_pars.prop_ph[0],
			                           pars.prop.size() * sizeof(std::complex<PRISM_FLOAT_PRECISION>),
			                           hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qxInd_d[g], &cuda_pars.qxInd_ph[0],
			                           pars.qxInd.size() * sizeof(size_t), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.qyInd_d[g], &cuda_pars.qyInd_ph[0],
			                           pars.qyInd.size() * sizeof(size_t), hipMemcpyHostToDevice, cuda_pars.streams[stream_id]));
			stream_id = (stream_id + pars.meta.NUM_GPUS) % total_num_streams;
			cudaErrchk(hipMemcpyAsync(cuda_pars.beamsIndex_d[g], &cuda_pars.beamsIndex_ph[0],
			                           pars.beamsIndex.size() * sizeof(size_t), hipMemcpyHostToDevice,
			                           cuda_pars.streams[stream_id]));
		}

		// make sure transfers are complete
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			hipSetDevice(g);
			hipDeviceSynchronize();
		}
	}
	inline void launchWorkers_singlexfer2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                              CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		// launch GPU work
		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations
		int stream_count = 0;
		const size_t PRISM_PRINT_FREQUENCY_BEAMS = max((size_t)1,pars.numberBeams / 10); // for printing status
		WorkDispatcher dispatcher(0, pars.numberBeams);
		for (auto t = 0; t < total_num_streams; ++t) {

			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipSetDevice(GPU_num);
			hipStream_t &current_stream = cuda_pars.streams[stream_count];
			cout << "Launching GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << endl;

			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_d = cuda_pars.trans_d[GPU_num];
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d = cuda_pars.prop_d[GPU_num];
			size_t *current_qxInd_d = cuda_pars.qxInd_d[GPU_num];
			size_t *current_qyInd_d = cuda_pars.qyInd_d[GPU_num];
			size_t *current_beamsIndex = cuda_pars.beamsIndex_d[GPU_num];
			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds = cuda_pars.psi_ds[stream_count];
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_small_ds = cuda_pars.psi_small_ds[stream_count];
			hipfftHandle &current_cufft_plan = cuda_pars.cufft_plans[stream_count];
			hipfftHandle &current_cufft_plan_small = cuda_pars.cufft_plans_small[stream_count];
			complex<PRISM_FLOAT_PRECISION> *current_S_slice_ph = cuda_pars.Scompact_slice_ph[stream_count];

			workers_GPU.push_back(thread([&pars, current_trans_d, current_prop_d, current_qxInd_d, current_qyInd_d, &dispatcher,
					                             current_psi_ds, current_psi_small_ds, &current_cufft_plan, &current_cufft_plan_small,
					                             current_S_slice_ph, current_beamsIndex, GPU_num, stream_count, &current_stream, &PRISM_PRINT_FREQUENCY_BEAMS]() {
				cudaErrchk(hipSetDevice(GPU_num));

#ifndef NDEBUG
				{
//					 check memory usage on the GPU
					std::lock_guard<mutex> lock(PRISM::mem_lock);
					size_t free_mem, total_mem;
					free_mem=total_mem=0;
					cudaErrchk(hipMemGetInfo(&free_mem, &total_mem));
					pars.max_mem = std::max(total_mem - free_mem, pars.max_mem);
//					cout << "max_mem = " << pars.max_mem << endl;
				}
#endif // NDEBUG

				size_t currentBeam, stopBeam;
				currentBeam=stopBeam=0;
//				while (getWorkID(pars, currentBeam, stopBeam)) {
				while (dispatcher.getWork(currentBeam, stopBeam, pars.meta.batch_size_GPU)) {
					while (currentBeam < stopBeam) {
						if (currentBeam % PRISM_PRINT_FREQUENCY_BEAMS < pars.meta.batch_size_GPU | currentBeam == 100){
							cout << "Computing Plane Wave #" << currentBeam << "/" << pars.numberBeams << endl;
						}
//						propagatePlaneWave_GPU_singlexfer(pars,
//						                                  current_trans_d,
//						                                  current_psi_ds,
//						                                  current_psi_small_ds,
//						                                  current_S_slice_ph,
//						                                  current_qyInd_d,
//						                                  current_qxInd_d,
//						                                  current_prop_d,
//						                                  current_beamsIndex,
//						                                  currentBeam,
//						                                  current_cufft_plan,
//						                                  current_cufft_plan_small,
//						                                  current_stream);
						propagatePlaneWave_GPU_singlexfer_batch(pars,
						                                        current_trans_d,
						                                        current_psi_ds,
						                                        current_psi_small_ds,
						                                        current_S_slice_ph,
						                                        current_qyInd_d,
						                                        current_qxInd_d,
						                                        current_prop_d,
						                                        current_beamsIndex,
						                                        currentBeam,
						                                        stopBeam,
						                                        current_cufft_plan,
						                                        current_cufft_plan_small,
						                                        current_stream);
#ifdef PRISM_BUILDING_GUI
						pars.progressbar->signalScompactUpdate(currentBeam, pars.numberBeams);
#endif
//						++currentBeam;
						currentBeam=stopBeam;
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << "finished\n";
			}));
			++stream_count;
		}

		if (pars.meta.also_do_CPU_work){

			// launch CPU work
			vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations
			mutex fftw_plan_lock;
			pars.meta.batch_size_CPU = min(pars.meta.batch_size_target_CPU, max((size_t)1, pars.numberBeams / pars.meta.NUM_THREADS));
			cout << "PRISM02 pars.meta.batch_size_CPU = " << pars.meta.batch_size_CPU << endl;
			PRISM_FFTW_INIT_THREADS();
			PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching thread #" << t << " to compute beams\n";
				workers_CPU.push_back(thread([&pars, &fftw_plan_lock, &dispatcher, &PRISM_PRINT_FREQUENCY_BEAMS]() {

					size_t currentBeam, stopBeam, early_CPU_stop;
					currentBeam=stopBeam=0;
					if (pars.meta.NUM_GPUS > 0){
						// if there are no GPUs, make sure to do all work on CPU
//						early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0,pars.numberBeams - pars.meta.gpu_cpu_ratio);
						early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0,pars.numberBeams - pars.meta.gpu_cpu_ratio * pars.meta.batch_size_CPU);
					} else {
						early_CPU_stop = pars.numberBeams;
					}
					if (dispatcher.getWork(currentBeam, stopBeam, pars.meta.batch_size_CPU, early_CPU_stop)) {
						// allocate array for psi just once per thread
						Array1D<complex<PRISM_FLOAT_PRECISION> > psi_stack = zeros_ND<1, complex<PRISM_FLOAT_PRECISION> >(
								{{pars.imageSize[0]*pars.imageSize[1]*pars.meta.batch_size_CPU}});

//						 setup batch FFTW parameters
						const int rank = 2;
						int n[] = {(int)pars.imageSize[0], (int)pars.imageSize[1]};
						const int howmany = pars.meta.batch_size_CPU;
						int idist = n[0]*n[1];
						int odist = n[0]*n[1];
						int istride = 1;
						int ostride = 1;
						int *inembed = n;
						int *onembed = n;



						unique_lock<mutex> gatekeeper(fftw_plan_lock);
						PRISM_FFTW_PLAN plan_forward = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_FORWARD, FFTW_MEASURE);
						PRISM_FFTW_PLAN plan_inverse = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_BACKWARD, FFTW_MEASURE);

						gatekeeper.unlock(); // unlock it so we only block as long as necessary to deal with plans

						// main work loop
						do { // synchronously get work assignment
							while (currentBeam < stopBeam) {
								if (currentBeam % PRISM_PRINT_FREQUENCY_BEAMS < pars.meta.batch_size_CPU | currentBeam == 100){
									cout << "Computing Plane Wave #" << currentBeam << "/" << pars.numberBeams << endl;
								}
								// re-zero psi each iteration
								memset((void *) &psi_stack[0], 0, psi_stack.size() * sizeof(complex<PRISM_FLOAT_PRECISION>));
//								propagatePlaneWave_CPU(pars, currentBeam, psi, plan_forward, plan_inverse, fftw_plan_lock);
								propagatePlaneWave_CPU_batch(pars, currentBeam, stopBeam, psi_stack, plan_forward, plan_inverse, fftw_plan_lock);
#ifdef PRISM_BUILDING_GUI
								pars.progressbar->signalScompactUpdate(currentBeam, pars.numberBeams);
#endif
//                                currentBeam = stopBeam;
								++currentBeam;
							}
							if (currentBeam >= early_CPU_stop) break;
						} while (dispatcher.getWork(currentBeam, stopBeam, pars.meta.batch_size_CPU, early_CPU_stop));
						// clean up
						gatekeeper.lock();
						PRISM_FFTW_DESTROY_PLAN(plan_forward);
						PRISM_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
				}));
			}
			for (auto &t:workers_CPU)t.join();
			PRISM_FFTW_CLEANUP_THREADS();
		}

		for (auto &t:workers_GPU)t.join();
	}
	 void launchWorkers_streaming(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                    CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		// launch GPU work
		vector<thread> workers_GPU;
		workers_GPU.reserve(total_num_streams); // prevents multiple reallocations
		int stream_count = 0;
		const size_t PRISM_PRINT_FREQUENCY_BEAMS = max((size_t)1,pars.numberBeams / 10); // for printing status
		WorkDispatcher dispatcher(0, pars.numberBeams);
		for (auto t = 0; t < total_num_streams; ++t) {
			int GPU_num = stream_count % pars.meta.NUM_GPUS; // determine which GPU handles this job
			hipSetDevice(GPU_num);
			hipStream_t &current_stream = cuda_pars.streams[stream_count];
			// get pointers to the pre-copied arrays, making sure to get those on the current GPU
			PRISM_CUDA_COMPLEX_FLOAT *current_prop_d = cuda_pars.prop_d[GPU_num];
			size_t *current_qxInd_d = cuda_pars.qxInd_d[GPU_num];
			size_t *current_qyInd_d = cuda_pars.qyInd_d[GPU_num];
			size_t *current_beamsIndex = cuda_pars.beamsIndex_d[GPU_num];
			// get pointers to per-stream arrays
			PRISM_CUDA_COMPLEX_FLOAT *current_trans_ds = cuda_pars.trans_d[stream_count];
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_ds = cuda_pars.psi_ds[stream_count];
			PRISM_CUDA_COMPLEX_FLOAT *current_psi_small_ds = cuda_pars.psi_small_ds[stream_count];
			hipfftHandle &current_cufft_plan = cuda_pars.cufft_plans[stream_count];
			hipfftHandle &current_cufft_plan_small = cuda_pars.cufft_plans_small[stream_count];
			complex<PRISM_FLOAT_PRECISION> *current_S_slice_ph = cuda_pars.Scompact_slice_ph[stream_count];

			workers_GPU.push_back(thread([&pars, current_trans_ds, current_prop_d, current_qxInd_d, current_qyInd_d, &dispatcher,
					                             current_psi_ds, current_psi_small_ds, &current_cufft_plan, &current_cufft_plan_small,
					                             current_S_slice_ph, current_beamsIndex, GPU_num, stream_count, &current_stream, &PRISM_PRINT_FREQUENCY_BEAMS, &cuda_pars]() {
				cudaErrchk(hipSetDevice(GPU_num));

#ifndef NDEBUG
				{
//					 check memory usage on the GPU
					std::lock_guard<mutex> lock(PRISM::mem_lock);
					size_t free_mem, total_mem;
					free_mem=total_mem=0;
					cudaErrchk(hipMemGetInfo(&free_mem, &total_mem));
					pars.max_mem = std::max(total_mem - free_mem, pars.max_mem);
					cout << "max_mem = " << pars.max_mem << endl;
				}
#endif // NDEBUG

				size_t currentBeam, stopBeam;
				currentBeam=stopBeam=0;
//				while (getWorkID(pars, currentBeam, stopBeam)) {
				while (dispatcher.getWork(currentBeam, stopBeam, pars.meta.batch_size_GPU)) {
					while (currentBeam < stopBeam) {
						if (currentBeam % PRISM_PRINT_FREQUENCY_BEAMS < pars.meta.batch_size_GPU | currentBeam == 100){
							cout << "Computing Plane Wave #" << currentBeam << "/" << pars.numberBeams << endl;
						}
//						propagatePlaneWave_GPU_streaming(pars,
//						                                 current_trans_ds,
//						                                 trans_ph,
//						                                 current_psi_ds,
//						                                 current_psi_small_ds,
//						                                 current_S_slice_ph,
//						                                 current_qyInd_d,
//						                                 current_qxInd_d,
//						                                 current_prop_d,
//						                                 current_beamsIndex,
//						                                 currentBeam,
//						                                 current_cufft_plan,
//						                                 current_cufft_plan_small,
//						                                 current_stream);
						propagatePlaneWave_GPU_streaming_batch(pars,
						                                       current_trans_ds,
						                                       cuda_pars.trans_ph,
						                                       current_psi_ds,
						                                       current_psi_small_ds,
						                                       current_S_slice_ph,
						                                       current_qyInd_d,
						                                       current_qxInd_d,
						                                       current_prop_d,
						                                       current_beamsIndex,
						                                       currentBeam,
						                                       stopBeam,
						                                       current_cufft_plan,
						                                       current_cufft_plan_small,
						                                       current_stream);
//						++currentBeam;
						currentBeam=stopBeam;
#ifdef PRISM_BUILDING_GUI
						pars.progressbar->signalScompactUpdate(currentBeam, pars.numberBeams);
#endif
					}
				}
				cout << "GPU worker on stream #" << stream_count << " of GPU #" << GPU_num << " finished\n";
			}));
			++stream_count;
		}

		if (pars.meta.also_do_CPU_work){

			// launch CPU work
			vector<thread> workers_CPU;
			workers_CPU.reserve(pars.meta.NUM_THREADS); // prevents multiple reallocations
			mutex fftw_plan_lock;
			pars.meta.batch_size_CPU = min(pars.meta.batch_size_target_CPU, max((size_t)1, pars.numberBeams / pars.meta.NUM_THREADS));
			cout << "PRISM02 pars.meta.batch_size_CPU = " << pars.meta.batch_size_CPU << endl;
			PRISM_FFTW_INIT_THREADS();
			PRISM_FFTW_PLAN_WITH_NTHREADS(pars.meta.NUM_THREADS);
			for (auto t = 0; t < pars.meta.NUM_THREADS; ++t) {
				cout << "Launching thread #" << t << " to compute beams\n";
				workers_CPU.push_back(thread([&pars, &fftw_plan_lock, &dispatcher, &PRISM_PRINT_FREQUENCY_BEAMS]() {

					size_t currentBeam, stopBeam, early_CPU_stop;
					currentBeam=stopBeam=0;
					if (pars.meta.NUM_GPUS > 0){
						// if there are no GPUs, make sure to do all work on CPU
						//early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0,pars.numberBeams - pars.meta.gpu_cpu_ratio);
						early_CPU_stop = (size_t)std::max((PRISM_FLOAT_PRECISION)0.0,pars.numberBeams - pars.meta.gpu_cpu_ratio*pars.meta.batch_size_CPU);
					} else {
						early_CPU_stop = pars.numberBeams;
					}
					if (dispatcher.getWork(currentBeam, stopBeam, pars.meta.batch_size_CPU, early_CPU_stop)) {
						// allocate array for psi just once per thread
//						Array2D<complex<PRISM_FLOAT_PRECISION> > psi = zeros_ND<2, complex<PRISM_FLOAT_PRECISION> >(
//								{{pars.imageSize[0], pars.imageSize[1]}});
						Array1D<complex<PRISM_FLOAT_PRECISION> > psi_stack = zeros_ND<1, complex<PRISM_FLOAT_PRECISION> >(
								{{pars.imageSize[0]*pars.imageSize[1]*pars.meta.batch_size_CPU}});

						// setup batch FFTW parameters
						const int rank = 2;
						int n[] = {(int)pars.imageSize[0], (int)pars.imageSize[1]};
						const int howmany = pars.meta.batch_size_CPU;
						int idist = n[0]*n[1];
						int odist = n[0]*n[1];
						int istride = 1;
						int ostride = 1;
						int *inembed = n;
						int *onembed = n;

						unique_lock<mutex> gatekeeper(fftw_plan_lock);
						PRISM_FFTW_PLAN plan_forward = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_FORWARD, FFTW_MEASURE);
						PRISM_FFTW_PLAN plan_inverse = PRISM_FFTW_PLAN_DFT_BATCH(rank, n, howmany,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), inembed,
						                                                         istride, idist,
						                                                         reinterpret_cast<PRISM_FFTW_COMPLEX *>(&psi_stack[0]), onembed,
						                                                         ostride, odist,
						                                                         FFTW_BACKWARD, FFTW_MEASURE);

						gatekeeper.unlock(); // unlock it so we only block as long as necessary to deal with plans

						// main work loop
						do { // synchronously get work assignment
							while (currentBeam < stopBeam) {
								if (currentBeam % PRISM_PRINT_FREQUENCY_BEAMS < pars.meta.batch_size_CPU | currentBeam == 100){
									cout << "Computing Plane Wave #" << currentBeam << "/" << pars.numberBeams << endl;
								}
								// re-zero psi each iteration
								memset((void *) &psi_stack[0], 0, psi_stack.size() * sizeof(complex<PRISM_FLOAT_PRECISION>));
//								propagatePlaneWave_CPU(pars, currentBeam, psi, plan_forward, plan_inverse, fftw_plan_lock);
								propagatePlaneWave_CPU_batch(pars, currentBeam, stopBeam, psi_stack, plan_forward, plan_inverse, fftw_plan_lock);
#ifdef PRISM_BUILDING_GUI
								pars.progressbar->signalScompactUpdate(currentBeam, pars.numberBeams);
#endif
								currentBeam = stopBeam;
//								++currentBeam;
							}
							if (currentBeam >= early_CPU_stop) break;
						} while (dispatcher.getWork(currentBeam, stopBeam, pars.meta.batch_size_CPU, early_CPU_stop));
						// clean up
						gatekeeper.lock();
						PRISM_FFTW_DESTROY_PLAN(plan_forward);
						PRISM_FFTW_DESTROY_PLAN(plan_inverse);
						gatekeeper.unlock();
					}
				}));
			}
			for (auto &t:workers_CPU)t.join();
			PRISM_FFTW_CLEANUP_THREADS();
		}
		for (auto &t:workers_GPU)t.join();
	}

	inline void cleanupMemory2(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                          CudaParameters<PRISM_FLOAT_PRECISION> &cuda_pars){
		const int total_num_streams = pars.meta.NUM_GPUS * pars.meta.NUM_STREAMS_PER_GPU;
		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g) {
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipFree(cuda_pars.trans_d[g]));
			cudaErrchk(hipFree(cuda_pars.prop_d[g]));
			cudaErrchk(hipFree(cuda_pars.qxInd_d[g]));
			cudaErrchk(hipFree(cuda_pars.qyInd_d[g]));
			cudaErrchk(hipFree(cuda_pars.beamsIndex_d[g]));
		}

		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipSetDevice(s % pars.meta.NUM_GPUS));
			cudaErrchk(hipFree(cuda_pars.psi_ds[s]));
			cudaErrchk(hipFree(cuda_pars.psi_small_ds[s]));
			cufftErrchk(hipfftDestroy(cuda_pars.cufft_plans[s]));
			cufftErrchk(hipfftDestroy(cuda_pars.cufft_plans_small[s]));
		}

		// free pinned memory
		for (auto s = 0; s < total_num_streams; ++s) {
			cudaErrchk(hipHostFree(cuda_pars.Scompact_slice_ph[s]));
		}
		cudaErrchk(hipHostFree(cuda_pars.trans_ph));
		cudaErrchk(hipHostFree(cuda_pars.prop_ph));
		cudaErrchk(hipHostFree(cuda_pars.qxInd_ph));
		cudaErrchk(hipHostFree(cuda_pars.qyInd_ph));
		cudaErrchk(hipHostFree(cuda_pars.beamsIndex_ph));


		// destroy CUDA streams
		for (auto j = 0; j < total_num_streams; ++j){
			hipSetDevice(j % pars.meta.NUM_GPUS);
			cudaErrchk(hipStreamDestroy(cuda_pars.streams[j]));
		}

		for (auto g = 0; g < pars.meta.NUM_GPUS; ++g){
			cudaErrchk(hipSetDevice(g));
			cudaErrchk(hipDeviceReset());
		}

		delete[] cuda_pars.streams;
		delete[] cuda_pars.cufft_plans;
		delete[] cuda_pars.cufft_plans_small;
		delete[] cuda_pars.trans_d;
		delete[] cuda_pars.prop_d;
		delete[] cuda_pars.qxInd_d;
		delete[] cuda_pars.qyInd_d;
		delete[] cuda_pars.beamsIndex_d;
		delete[] cuda_pars.psi_ds;
		delete[] cuda_pars.psi_small_ds;
		delete[] cuda_pars.Scompact_slice_ph;
	}

	void propagatePlaneWave_GPU_singlexfer(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                       PRISM_CUDA_COMPLEX_FLOAT* trans_d,
		  	                               PRISM_CUDA_COMPLEX_FLOAT* psi_d,
			                               PRISM_CUDA_COMPLEX_FLOAT* psi_small_d,
			                               complex<PRISM_FLOAT_PRECISION>* Scompact_slice_ph,
			                               const size_t* qyInd_d,
			                               const size_t* qxInd_d,
			                               const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
			                               const size_t* beamsIndex,
			                               const size_t beamNumber,
			                               const hipfftHandle& plan,
			                               const hipfftHandle& plan_small,
			                               hipStream_t& stream){

		const size_t psi_size = pars.imageSize[0] * pars.imageSize[1];
		const size_t psi_small_size = pars.qxInd.size() * pars.qyInd.size();
		initializePsi_oneNonzero<<< (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(psi_d, psi_size, pars.beamsIndex[beamNumber]);


		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_BACKWARD));
			multiply_cx<<<(psi_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, &trans_d[planeNum*psi_size], psi_size);
			divide_inplace<<<(psi_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, PRISM_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_FORWARD));
			multiply_cx<<<(psi_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, prop_d, psi_size);
		}

		array_subset<<<(pars.qyInd.size()*pars.qxInd.size()-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>> (
		psi_d, psi_small_d, qyInd_d, qxInd_d, pars.imageSize[1], pars.qyInd.size(), pars.qxInd.size());

		PRISM_CUFFT_EXECUTE(plan_small,&psi_small_d[0], &psi_small_d[0], HIPFFT_BACKWARD);
        divide_inplace<<<(psi_small_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_small_d, PRISM_MAKE_CU_COMPLEX(psi_small_size, 0),psi_small_size);

		cudaErrchk(hipMemcpyAsync(Scompact_slice_ph,&psi_small_d[0],psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost,stream));
		hipStreamSynchronize(stream);
		memcpy(&pars.Scompact[beamNumber * pars.Scompact.get_dimj() * pars.Scompact.get_dimi()], &Scompact_slice_ph[0], psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT));

	}

	void propagatePlaneWave_GPU_singlexfer_batch(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                             PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                             PRISM_CUDA_COMPLEX_FLOAT* psi_d,
	                                             PRISM_CUDA_COMPLEX_FLOAT* psi_small_d,
	                                             complex<PRISM_FLOAT_PRECISION>* Scompact_slice_ph,
	                                             const size_t* qyInd_d,
	                                             const size_t* qxInd_d,
	                                             const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                             const size_t* beamsIndex,
	                                             const size_t beamNumber,
	                                             const size_t stopBeam,
	                                             const hipfftHandle& plan,
	                                             const hipfftHandle& plan_small,
	                                             hipStream_t& stream){

		const size_t psi_size        = pars.imageSize[0] * pars.imageSize[1];
		const size_t psi_small_size = pars.qxInd.size() * pars.qyInd.size();
		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
			// initialize psi
			initializePsi_oneNonzero<<< (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(psi_d + batch_idx*psi_size, psi_size, pars.beamsIndex[beamNumber + batch_idx]);
		}
		for (auto planeNum = 0; planeNum < pars.numPlanes; ++planeNum) {
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_BACKWARD));
			for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
				multiply_cx << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_d + batch_idx*psi_size, &trans_d[planeNum * psi_size], psi_size);
				divide_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>
						(psi_d + batch_idx*psi_size, PRISM_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
			}
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_FORWARD));
			for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
				multiply_cx << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> > (psi_d + batch_idx*psi_size, prop_d, psi_size);
			}
		}

		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
			array_subset << < (pars.qyInd.size() * pars.qxInd.size() - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0,
					stream >> > (psi_d + batch_idx*psi_size, psi_small_d + batch_idx*psi_small_size, qyInd_d, qxInd_d, pars.imageSize[1], pars.qyInd.size(), pars.qxInd.size());
		}

		PRISM_CUFFT_EXECUTE(plan_small,&psi_small_d[0], &psi_small_d[0], HIPFFT_BACKWARD);
		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
			divide_inplace<<<(psi_small_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_small_d + batch_idx*psi_small_size, PRISM_MAKE_CU_COMPLEX(psi_small_size, 0), psi_small_size);
		}
		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
			cudaErrchk(hipMemcpyAsync(Scompact_slice_ph, &psi_small_d[0 + batch_idx*psi_small_size],
			                           psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT),
			                           hipMemcpyDeviceToHost, stream));
			hipStreamSynchronize(stream);
			memcpy(&pars.Scompact[(beamNumber + batch_idx) * pars.Scompact.get_dimj() * pars.Scompact.get_dimi()],
			       &Scompact_slice_ph[0], psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT));
		}
	}

	void propagatePlaneWave_GPU_streaming(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                      PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                      const std::complex<PRISM_FLOAT_PRECISION> *trans_ph,
	                                      PRISM_CUDA_COMPLEX_FLOAT* psi_d,
	                                      PRISM_CUDA_COMPLEX_FLOAT* psi_small_d,
	                                      complex<PRISM_FLOAT_PRECISION>* Scompact_slice_ph,
	                                      const size_t* qyInd_d,
	                                      const size_t* qxInd_d,
	                                      const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                      const size_t* beamsIndex,
	                                      const size_t beamNumber,
	                                      const hipfftHandle& plan,
	                                      const hipfftHandle& plan_small,
	                                      hipStream_t& stream){
		// In this version, each slice of the transmission matrix is streamed to the device

		const size_t psi_size = pars.imageSize[0] * pars.imageSize[1];
		const size_t psi_small_size = pars.qxInd.size() * pars.qyInd.size();
		initializePsi_oneNonzero<<< (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(psi_d, psi_size, pars.beamsIndex[beamNumber]);

		for (auto planeNum = 0; planeNum < pars.numPlanes ; ++planeNum) {
			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*psi_size], psi_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_BACKWARD));
			multiply_cx<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, trans_d, psi_size);
			divide_inplace<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, PRISM_MAKE_CU_COMPLEX(psi_size, 0), psi_size);
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_FORWARD));
			multiply_cx<<<(psi_size - 1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_d, prop_d, psi_size);
		}
		array_subset<<<(pars.qyInd.size()*pars.qxInd.size()-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>> (
				psi_d, psi_small_d, qyInd_d, qxInd_d, pars.imageSize[1], pars.qyInd.size(), pars.qxInd.size());

		PRISM_CUFFT_EXECUTE(plan_small,&psi_small_d[0], &psi_small_d[0], HIPFFT_BACKWARD);
		divide_inplace<<<(psi_small_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>(psi_small_d, PRISM_MAKE_CU_COMPLEX(psi_small_size, 0),psi_small_size);

		cudaErrchk(hipMemcpyAsync(Scompact_slice_ph,&psi_small_d[0],psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost,stream));
		hipStreamSynchronize(stream);
		memcpy(&pars.Scompact[beamNumber * pars.Scompact.get_dimj() * pars.Scompact.get_dimi()], &Scompact_slice_ph[0], psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT));
	}


	void propagatePlaneWave_GPU_streaming_batch(Parameters<PRISM_FLOAT_PRECISION> &pars,
	                                            PRISM_CUDA_COMPLEX_FLOAT* trans_d,
	                                            const std::complex<PRISM_FLOAT_PRECISION> *trans_ph,
	                                            PRISM_CUDA_COMPLEX_FLOAT* psi_d,
	                                            PRISM_CUDA_COMPLEX_FLOAT* psi_small_d,
	                                            complex<PRISM_FLOAT_PRECISION>* Scompact_slice_ph,
	                                            const size_t* qyInd_d,
	                                            const size_t* qxInd_d,
	                                            const PRISM_CUDA_COMPLEX_FLOAT* prop_d,
	                                            const size_t* beamsIndex,
	                                            const size_t beamNumber,
	                                            const size_t stopBeam,
	                                            const hipfftHandle& plan,
	                                            const hipfftHandle& plan_small,
	                                            hipStream_t& stream){
		// In this version, each slice of the transmission matrix is streamed to the device

		const size_t psi_size        = pars.imageSize[0] * pars.imageSize[1];
		const size_t psi_small_size = pars.qxInd.size() * pars.qyInd.size();
		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
			// initialize psi -- for PRISM this is just a delta function in Fourier space located depending on which plane wave it is
			initializePsi_oneNonzero<<< (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream>>>(psi_d + batch_idx*psi_size, psi_size, pars.beamsIndex[beamNumber + batch_idx]);
		}

		for (auto planeNum = 0; planeNum < pars.numPlanes ; ++planeNum) {
			cudaErrchk(hipMemcpyAsync(trans_d, &trans_ph[planeNum*psi_size], psi_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT), hipMemcpyHostToDevice, stream));
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_BACKWARD));
			for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
				multiply_cx << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_d + batch_idx*psi_size, trans_d, psi_size); // transmit
				divide_inplace << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_d + batch_idx*psi_size, PRISM_MAKE_CU_COMPLEX(psi_size, 0), psi_size); // normalize the FFT
			}
			cufftErrchk(PRISM_CUFFT_EXECUTE(plan, &psi_d[0], &psi_d[0], HIPFFT_FORWARD));
			for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
				multiply_cx << < (psi_size - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >> >
						(psi_d + batch_idx*psi_size, prop_d, psi_size); // propagate
			}
		}

		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
			// take relevant subset of the full array
			array_subset << < (pars.qyInd.size() * pars.qxInd.size() - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0,
					stream >> > (psi_d + batch_idx*psi_size, psi_small_d + batch_idx*psi_small_size, qyInd_d, qxInd_d, pars.imageSize[1], pars.qyInd.size(), pars.qxInd.size());
		}

		// final FFT
		PRISM_CUFFT_EXECUTE(plan_small,&psi_small_d[0], &psi_small_d[0], HIPFFT_BACKWARD);
		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
		divide_inplace<<<(psi_small_size-1) / BLOCK_SIZE1D + 1,BLOCK_SIZE1D, 0, stream>>>
				(psi_small_d + batch_idx*psi_small_size, PRISM_MAKE_CU_COMPLEX(psi_small_size, 0),psi_small_size); // normalize the FFT
			}

		// copy the result
		for (auto batch_idx = 0; batch_idx < (stopBeam-beamNumber); ++batch_idx) {
		cudaErrchk(hipMemcpyAsync(Scompact_slice_ph,&psi_small_d[batch_idx*psi_small_size],psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT),hipMemcpyDeviceToHost,stream));
		hipStreamSynchronize(stream);
		memcpy(&pars.Scompact[beamNumber * pars.Scompact.get_dimj() * pars.Scompact.get_dimi()], &Scompact_slice_ph[0], psi_small_size * sizeof(PRISM_CUDA_COMPLEX_FLOAT));
			}
	}

	inline void setupArrays2(Parameters<PRISM_FLOAT_PRECISION>& pars){

		// setup some needed arrays
		const PRISM_FLOAT_PRECISION pi = acos(-1);
		const std::complex<PRISM_FLOAT_PRECISION> i(0, 1);
		pars.Scompact = zeros_ND<3, complex<PRISM_FLOAT_PRECISION> >(
				{{pars.numberBeams, pars.imageSize[0] / 2, pars.imageSize[1] / 2}});
		pars.transmission = zeros_ND<3, complex<PRISM_FLOAT_PRECISION> >(
				{{pars.pot.get_dimk(), pars.pot.get_dimj(), pars.pot.get_dimi()}});
		{
			auto p = pars.pot.begin();
			for (auto &j:pars.transmission)j = exp(i * pars.sigma * (*p++));
		}
	}

	void fill_Scompact_GPU_singlexfer(Parameters <PRISM_FLOAT_PRECISION> &pars) {

		// This version transfers the entire transmission matrix a single time, which results in faster execution but requires more memory
#ifdef PRISM_BUILDING_GUI
		pars.progressbar->signalDescriptionMessage("Computing compact S-matrix");
		pars.progressbar->signalScompactUpdate(-1, pars.numberBeams);
#endif
		CudaParameters<PRISM_FLOAT_PRECISION> cuda_pars;

		// determine the batch size to use
        pars.meta.batch_size_GPU = min(pars.meta.batch_size_target_GPU, max((size_t)1, pars.numberBeams / max((size_t)1,(pars.meta.NUM_STREAMS_PER_GPU*pars.meta.NUM_GPUS))));

		// setup some arrays
		setupArrays2(pars);

		// create CUDA streams
		createStreamsAndPlans2(pars, cuda_pars);

		// create page-locked (pinned) host memory buffers
		allocatePinnedHostMemory_singlexfer2(pars, cuda_pars);

		// copy to pinned memory
		copyToPinnedMemory_singlexfer2(pars, cuda_pars);

		// allocate memory on the GPUs
		allocateDeviceMemory_singlexfer2(pars, cuda_pars);

		// copy to GPUs
		copyToDeviceMemory_singlexfer2(pars, cuda_pars);

		// launch workers
		launchWorkers_singlexfer2(pars, cuda_pars);

		// free memory on the host/device
		cleanupMemory2(pars, cuda_pars);
	}

	void fill_Scompact_GPU_streaming(Parameters <PRISM_FLOAT_PRECISION> &pars) {

#ifdef PRISM_BUILDING_GUI
		pars.progressbar->signalDescriptionMessage("Computing compact S-matrix");
		pars.progressbar->signalScompactUpdate(-1, pars.numberBeams);
#endif
		// This version streams each slice of the transmission matrix, which is less efficient but can tolerate very large arrays
		//initialize data
		CudaParameters<PRISM_FLOAT_PRECISION> cuda_pars;

		// determine the batch size to use
		pars.meta.batch_size_GPU = min(pars.meta.batch_size_target_GPU, max((size_t)1, pars.numberBeams / max((size_t)1,(pars.meta.NUM_STREAMS_PER_GPU*pars.meta.NUM_GPUS))));

		// setup some arrays
		setupArrays2(pars);

		// create CUDA streams and cuFFT plans
		createStreamsAndPlans2(pars, cuda_pars);

		// create page-locked (pinned) host memory buffers
		allocatePinnedHostMemory_streaming2(pars, cuda_pars);

		// copy to pinned memory
		copyToPinnedMemory_streaming2(pars, cuda_pars);

		// allocate memory on the GPUs
		allocateDeviceMemory_streaming2(pars, cuda_pars);

		// copy to GPUs
		copyToDeviceMemory_streaming2(pars, cuda_pars);

		// launch workers
		launchWorkers_streaming(pars, cuda_pars);

		// free memory on the host/device
		cleanupMemory2(pars, cuda_pars);
	}
}