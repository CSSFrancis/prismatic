#include "hip/hip_runtime.h"
#include "utility.cuh"
#include <iostream>

#define PI 3.14159265359
// define some constants
__device__ __constant__ float pi_f       = PI;
__device__ __constant__ hipFloatComplex i_f     = {0, 1};
__device__ __constant__ hipFloatComplex pi_cx_f = {PI, 0};
__device__ __constant__ hipFloatComplex minus_2pii_f = {0, -2*PI};
__device__ __constant__ double pi       = PI;
__device__ __constant__ hipDoubleComplex i     = {0, 1};
__device__ __constant__ hipDoubleComplex pi_cx = {PI, 0};
__device__ __constant__ hipDoubleComplex minus_2pii = {0, -2*PI};

// computes exp(real(a) + i * imag(a))
__device__ __forceinline__ hipDoubleComplex exp_cx(const hipDoubleComplex a){
	double e = exp(a.x);
	double s,c;
	sincos(a.y, &s, &c);
	return make_hipDoubleComplex(e*c, e*s);
}
__device__ __forceinline__ hipFloatComplex exp_cx(const hipFloatComplex a){
	float e = expf(a.x);
	float s,c;
	sincosf(a.y, &s, &c);
	return make_hipFloatComplex(e*c, e*s);
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi_oneNonzero(hipFloatComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipFloatComplex(1,0):make_hipFloatComplex(0,0);
	}
}

__global__ void initializePsi_oneNonzero(hipDoubleComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipDoubleComplex(1,0):make_hipDoubleComplex(0,0);
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipDoubleComplex* arr,
                                 const hipDoubleComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex a = arr[idx];
		hipDoubleComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipFloatComplex* arr,
                                 const hipFloatComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex a = arr[idx];
		hipFloatComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipDoubleComplex* arr,
                             const hipDoubleComplex* other,
                             const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
//		hipDoubleComplex a = arr[idx];
//		hipDoubleComplex o = other[idx];
		arr[idx] = hipCmul(arr[idx], other[idx]);
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipFloatComplex* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
//		hipFloatComplex a = arr[idx];
//		hipFloatComplex o = other[idx];
		arr[idx] = hipCmulf(arr[idx], other[idx]);
	}
}

//// divide two complex arrays
//__global__ void divide_inplace(PRISM_CUDA_COMPLEX_FLOAT* arr,
//                               const PRISM_FLOAT_PRECISION val,
//                               const size_t N){
//	int idx = threadIdx.x + blockDim.x*blockIdx.x;
//	if (idx < N) {
//		arr[idx].x /= val;
//		arr[idx].y /= val;
//	}
//}

__global__ void divide_inplace(hipDoubleComplex* arr,
                               const hipDoubleComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdiv(arr[idx], val);
	}
}

__global__ void divide_inplace(hipFloatComplex* arr,
                               const hipFloatComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdivf(arr[idx], val);
	}
}

// set all array values to val
__global__ void setAll(double *data, double val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// set all array values to val
__global__ void setAll(float *data, float val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipDoubleComplex *psi_d,
                              const hipDoubleComplex* PsiProbeInit_d,
                              const double* qya_d,
                              const double* qxa_d,
                              const size_t N,
                              const double yp,
                              const double xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex arg;
		arg = make_hipDoubleComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmul(PsiProbeInit_d[idx], exp_cx(hipCmul(minus_2pii,arg)));
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipFloatComplex *psi_d,
                              const hipFloatComplex* PsiProbeInit_d,
                              const float* qya_d,
                              const float* qxa_d,
                              const size_t N,
                              const float yp,
                              const float xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex arg;
		arg = make_hipFloatComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmulf(PsiProbeInit_d[idx], exp_cx(hipCmulf(minus_2pii_f,arg)));
	}
}


// compute modulus squared of other and store in arr
__global__ void abs_squared(double* arr,
                            const hipDoubleComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		double re = other[idx].x;
		double im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void abs_squared(float* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		float re = other[idx].x;
		float im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

__global__ void array_subset(const hipDoubleComplex* psi_d,
                             hipDoubleComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < dimj_small*dimi_small) {
		int y = idx / (int)dimi_small;
		int x = idx % (int)dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}
__global__ void array_subset(const hipFloatComplex* psi_d,
                             hipFloatComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < dimj_small*dimi_small) {
		int y = idx / (int)dimi_small;
		int x = idx % (int)dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}